#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/tuple.h>

#include <cstddef>
#include <numeric>
#include <optional>
#include <stdexcept>

#include "shuffle_split.hpp"

namespace spark_rapids_jni {

namespace {

using namespace cudf;

// The size that contiguous split uses internally as the GPU unit of work.
// The number of `desired_batch_size` batches equals the number of CUDA blocks
// that will be used for the main kernel launch (`copy_partitions`).
constexpr std::size_t desired_batch_size = 1 * 1024 * 1024;

// there will only be one copy of this 
struct cz_metadata_internal {
  // size_type                         num_columns = 0;
  size_type                         per_partition_metadata_size = 0;
  size_type                         max_depth = 0;
  shuffle_split_metadata            global_metadata;
};

/**
 * @brief Struct which contains information on a source buffer.
 *
 * The definition of "buffer" used throughout this module is a component piece of a
 * cudf column. So for example, a fixed-width column with validity would have 2 associated
 * buffers : the data itself and the validity buffer.  contiguous_split operates by breaking
 * each column up into it's individual components and copying each one as a separate kernel
 * block.
 */
struct src_buf_info {
  src_buf_info(cudf::type_id _type,
               int const* _offsets,
               int _offset_stack_pos,
               int _parent_offsets_index,
               bool _is_validity,
               size_type _column_offset)
    : type(_type),
      offsets(_offsets),
      offset_stack_pos(_offset_stack_pos),
      parent_offsets_index(_parent_offsets_index),
      is_validity(_is_validity),
      column_offset(_column_offset)
  {
  }

  src_buf_info(){}

  cudf::type_id type;
  int const* offsets;        // a pointer to device memory offsets if I am an offset buffer
  int offset_stack_pos;      // position in the offset stack buffer
  int parent_offsets_index;  // immediate parent that has offsets, or -1 if none
  bool is_validity;          // if I am a validity buffer
  size_type column_offset;   // offset in the case of a sliced column
};

/**
 * @brief Struct which contains information on a destination buffer.
 *
 * Similar to src_buf_info, dst_buf_info contains information on a destination buffer we
 * are going to copy to.  If we have N input buffers (which come from X columns), and
 * M partitions, then we have N*M destination buffers.
 */
struct dst_buf_info {
  // constant across all copy commands for this buffer
  std::size_t buf_size;  // total size of buffer, including padding
  int num_elements;      // # of elements to be copied
  int element_size;      // size of each element in bytes
  int num_rows;          // # of rows to be copied(which may be different from num_elements in the case of
                         // validity or offset buffers)

  int src_element_index;   // element index to start reading from my associated source buffer
  std::size_t dst_offset;  // my offset into the per-partition allocation, not including the per-partition metadata header size
  int value_shift;         // amount to shift values down by (for offset buffers)
  int bit_shift;           // # of bits to shift right by (for validity buffers)
  size_type valid_count;   // validity count for this block of work

  int src_buf_index;       // source buffer index
  int root_num_rows;     // for string columns, num_rows will be the number of chars. root_num_rows will be the number of top level rows
};

constexpr size_t size_to_batch_count(size_t bytes)
{
  return std::max(std::size_t{1}, util::round_up_unsafe(bytes, desired_batch_size) / desired_batch_size);
}

/**
 * @brief Copy a single buffer of column data, shifting values (for offset columns),
 * and validity (for validity buffers) as necessary.
 *
 * Copies a single partition of a source column buffer to a destination buffer. Shifts
 * element values by value_shift in the case of a buffer of offsets (value_shift will
 * only ever be > 0 in that case).  Shifts elements bitwise by bit_shift in the case of
 * a validity buffer (bit_shift will only ever be > 0 in that case).  This function assumes
 * value_shift and bit_shift will never be > 0 at the same time.
 *
 * This function expects:
 * - src may be a misaligned address
 * - dst must be an aligned address
 *
 * This function always does the ALU work related to value_shift and bit_shift because it is
 * entirely memory-bandwidth bound.
 *
 * @param dst Destination buffer
 * @param src Source buffer
 * @param t Thread index
 * @param num_elements Number of elements to copy
 * @param element_size Size of each element in bytes
 * @param src_element_index Element index to start copying at
 * @param stride Size of the kernel block
 * @param value_shift Shift incoming 4-byte offset values down by this amount
 * @param bit_shift Shift incoming data right by this many bits
 * @param num_rows Number of rows being copied
 * @param valid_count Optional pointer to a value to store count of set bits
 */
template <int block_size>
__device__ void copy_buffer(uint8_t* __restrict__ dst,
                            uint8_t const* __restrict__ src,
                            int t,
                            std::size_t num_elements,
                            std::size_t element_size,
                            std::size_t src_element_index,
                            uint32_t stride,
                            int value_shift,
                            int bit_shift,
                            std::size_t num_rows,
                            size_type* valid_count)
{
  src += (src_element_index * element_size);

  size_type thread_valid_count = 0;

  // handle misalignment. read 16 bytes in 4 byte reads. write in a single 16 byte store.
  std::size_t const num_bytes = num_elements * element_size;
  // how many bytes we're misaligned from 4-byte alignment
  uint32_t const ofs = reinterpret_cast<uintptr_t>(src) % 4;
  std::size_t pos    = t * 16;
  stride *= 16;
  while (pos + 20 <= num_bytes) {
    // read from the nearest aligned address.
    uint32_t const* in32 = reinterpret_cast<uint32_t const*>((src + pos) - ofs);
    uint4 v              = uint4{in32[0], in32[1], in32[2], in32[3]};
    if (ofs || bit_shift) {
      v.x = __funnelshift_r(v.x, v.y, ofs * 8 + bit_shift);
      v.y = __funnelshift_r(v.y, v.z, ofs * 8 + bit_shift);
      v.z = __funnelshift_r(v.z, v.w, ofs * 8 + bit_shift);
      v.w = __funnelshift_r(v.w, in32[4], ofs * 8 + bit_shift);
    }
    v.x -= value_shift;
    v.y -= value_shift;
    v.z -= value_shift;
    v.w -= value_shift;
    reinterpret_cast<uint4*>(dst)[pos / 16] = v;
    if (valid_count) {
      thread_valid_count += (__popc(v.x) + __popc(v.y) + __popc(v.z) + __popc(v.w));
    }
    pos += stride;
  }

  // copy trailing bytes
  if (t == 0) {
    std::size_t remainder;
    if (num_bytes < 16) {
      remainder = num_bytes;
    } else {
      std::size_t const last_bracket = (num_bytes / 16) * 16;
      remainder                      = num_bytes - last_bracket;
      if (remainder < 4) {
        // we had less than 20 bytes for the last possible 16 byte copy, so copy 16 + the extra
        remainder += 16;
      }
    }

    // if we're performing a value shift (offsets), or a bit shift (validity) the # of bytes and
    // alignment must be a multiple of 4. value shifting and bit shifting are mutually exclusive
    // and will never both be true at the same time.
    if (value_shift || bit_shift) {
      std::size_t idx = (num_bytes - remainder) / 4;
      uint32_t v = remainder > 0 ? (reinterpret_cast<uint32_t const*>(src)[idx] - value_shift) : 0;

      constexpr size_type rows_per_element = 32;
      auto const have_trailing_bits = ((num_elements * rows_per_element) - num_rows) < bit_shift;
      while (remainder) {
        // if we're at the very last word of a validity copy, we do not always need to read the next
        // word to get the final trailing bits.
        auto const read_trailing_bits = bit_shift > 0 && remainder == 4 && have_trailing_bits;
        uint32_t const next           = (read_trailing_bits || remainder > 4)
                                          ? (reinterpret_cast<uint32_t const*>(src)[idx + 1] - value_shift)
                                          : 0;

        uint32_t const val = (v >> bit_shift) | (next << (32 - bit_shift));
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint32_t*>(dst)[idx] = val;
        v                                     = next;
        idx++;
        remainder -= 4;
      }
    } else {
      while (remainder) {
        std::size_t const idx = num_bytes - remainder--;
        uint32_t const val    = reinterpret_cast<uint8_t const*>(src)[idx];
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint8_t*>(dst)[idx] = val;
      }
    }
  }

  if (valid_count) {
    if (num_bytes == 0) {
      if (!t) { *valid_count = 0; }
    } else {
      using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      size_type block_valid_count{BlockReduce(temp_storage).Sum(thread_valid_count)};
      if (!t) {
        // we may have copied more bits than there are actual rows in the output.
        // so we need to subtract off the count of any bits that shouldn't have been
        // considered during the copy step.
        std::size_t const max_row    = (num_bytes * 8);
        std::size_t const slack_bits = max_row > num_rows ? max_row - num_rows : 0;
        auto const slack_mask        = set_most_significant_bits(slack_bits);
        if (slack_mask > 0) {
          uint32_t const last_word = reinterpret_cast<uint32_t*>(dst + (num_bytes - 4))[0];
          block_valid_count -= __popc(last_word & slack_mask);
        }
        *valid_count = block_valid_count;
      }
    }
  }
}

/**
 * @brief Kernel which copies data from multiple source buffers to multiple
 * destination buffers.
 *
 * When doing a contiguous_split on X columns comprising N total internal buffers
 * with M splits, we end up having to copy N*M source/destination buffer pairs.
 * These logical copies are further subdivided to distribute the amount of work
 * to be done as evenly as possible across the multiprocessors on the device.
 * This kernel is arranged such that each block copies 1 source/destination pair.
 *
 * @param index_to_buffer A function that given a `buf_index` returns the destination buffer
 * @param src_bufs Input source buffers
 * @param buf_info Information on the range of values to be copied for each destination buffer
 */
template <int block_size, typename IndexToDstBuf>
CUDF_KERNEL void copy_partitions(IndexToDstBuf index_to_buffer,
                                 uint8_t const** src_bufs,
                                 dst_buf_info* buf_info)
{
  auto const buf_index     = blockIdx.x;
  auto const src_buf_index = buf_info[buf_index].src_buf_index;

  /*
  if(threadIdx.x == 0){
    printf("buf_index = %d, src_buf_index = %d, offset = %lu\n", (int)buf_index, (int)src_buf_index, (size_t)(buf_info[buf_index].dst_offset));
  }
  */

  // copy, shifting offsets and validity bits as needed
  copy_buffer<block_size>(
    // each buffer has a block of metadata at the very beginning that we need to skip past
    index_to_buffer(buf_index) + buf_info[buf_index].dst_offset,
    src_bufs[src_buf_index],
    threadIdx.x,
    buf_info[buf_index].num_elements,
    buf_info[buf_index].element_size,
    buf_info[buf_index].src_element_index,
    blockDim.x,
    buf_info[buf_index].value_shift,
    buf_info[buf_index].bit_shift,
    buf_info[buf_index].num_rows,
    buf_info[buf_index].valid_count > 0 ? &buf_info[buf_index].valid_count : nullptr);

  /*
  if(threadIdx.x == 0){
    printf("V(%d): %d\n", (int)buf_index, ((int*)(index_to_buffer(buf_index) + buf_info[buf_index].dst_offset))[0]);
  }
  */
}

// The block of functions below are all related:
//
// compute_offset_stack_size()
// setup_src_buf_data()
// count_src_bufs()
// setup_source_buf_info()
// build_output_columns()
//
// Critically, they all traverse the hierarchy of source columns and their children
// in a specific order to guarantee they produce various outputs in a consistent
// way.  For example, setup_src_buf_info() produces a series of information
// structs that must appear in the same order that setup_src_buf_data() produces
// buffers.
//
// So please be careful if you change the way in which these functions and
// functors traverse the hierarchy.

/**
 * @brief Returns whether or not the specified type is a column that contains offsets.
 */
bool is_offset_type(type_id id) { return (id == type_id::STRING or id == type_id::LIST); }

/**
 * @brief Compute total device memory stack size needed to process nested
 * offsets per-output buffer.
 *
 * When determining the range of rows to be copied for each output buffer
 * we have to recursively apply the stack of offsets from our parent columns
 * (lists or strings).  We want to do this computation on the gpu because offsets
 * are stored in device memory.  However we don't want to do recursion on the gpu, so
 * each destination buffer gets a "stack" of space to work with equal in size to
 * it's offset nesting depth.  This function computes the total size of all of those
 * stacks.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param offset_depth Current offset nesting depth
 *
 * @returns Total offset stack size needed for this range of columns
 */
template <typename InputIter>
std::size_t compute_offset_stack_size(InputIter begin, InputIter end, int offset_depth = 0)
{
  return std::accumulate(begin, end, 0, [offset_depth](auto stack_size, column_view const& col) {
    auto const num_buffers = 1 + (col.nullable() ? 1 : 0);
    return stack_size + (offset_depth * num_buffers) +
           compute_offset_stack_size(
             col.child_begin(), col.child_end(), offset_depth + is_offset_type(col.type().id()));
  });
}

/**
 * @brief Retrieve all buffers for a range of source columns.
 *
 * Retrieve the individual buffers that make up a range of input columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param out_buf Iterator into output buffer infos
 *
 * @returns next output buffer iterator
 */
template <typename InputIter, typename OutputIter>
OutputIter setup_src_buf_data(InputIter begin, InputIter end, OutputIter out_buf)
{
  std::for_each(begin, end, [&out_buf](column_view const& col) {
    if (col.nullable()) {
      *out_buf = reinterpret_cast<uint8_t const*>(col.null_mask());
      out_buf++;
    }
    // NOTE: we're always returning the base pointer here. column-level offset is accounted
    // for later. Also, for some column types (string, list, struct) this pointer will be null
    // because there is no associated data with the root column.
    *out_buf = col.head<uint8_t>();
    out_buf++;

    out_buf = setup_src_buf_data(col.child_begin(), col.child_end(), out_buf);
  });
  return out_buf;
}

/**
 * @brief Count the total number of source buffers we will be copying
 * from.
 *
 * This count includes buffers for all input columns. For example a
 * fixed-width column with validity would be 2 buffers (data, validity).
 * A string column with validity would be 3 buffers (chars, offsets, validity).
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 *
 * @returns total number of source buffers for this range of columns
 */
template <typename InputIter>
size_type count_src_bufs(InputIter begin, InputIter end)
{
  auto buf_iter = thrust::make_transform_iterator(begin, [](column_view const& col) {
    auto const children_counts = count_src_bufs(col.child_begin(), col.child_end());
    return 1 + (col.nullable() ? 1 : 0) + children_counts;
  });
  return std::accumulate(buf_iter, buf_iter + std::distance(begin, end), 0);
}

/**
 * @brief Computes source buffer information for the copy kernel.
 *
 * For each input column to be split we need to know several pieces of information
 * in the copy kernel.  This function traverses the input columns and prepares this
 * information for the gpu.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param head Beginning of source buffer info array
 * @param current Current source buffer info to be written to
 * @param offset_stack_pos Integer representing our current offset nesting depth
 * (how many list or string levels deep we are)
 * @param parent_offset_index Index into src_buf_info output array indicating our nearest
 * containing list parent. -1 if we have no list parent
 * @param offset_depth Current offset nesting depth (how many list levels deep we are)
 *
 * @returns next src_buf_output after processing this range of input columns
 */
// setup source buf info
template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          std::vector<size_type>& metadata_col_to_buf_index,
                                                          rmm::cuda_stream_view stream,
                                                          int offset_stack_pos    = 0,
                                                          int parent_offset_index = -1,
                                                          int offset_depth        = 0);

/**
 * @brief Functor that builds source buffer information based on input columns.
 *
 * Called by setup_source_buf_info to build information for a single source column.  This function
 * will recursively call setup_source_buf_info in the case of nested types.
 */
struct buf_info_functor {
  src_buf_info* head;

  template <typename T>
  std::pair<src_buf_info*, size_type> operator()(column_view const& col,
                                                 src_buf_info* current,
                                                 int offset_stack_pos,
                                                 int parent_offset_index,
                                                 int offset_depth,
                                                 std::vector<size_type>& metadata_col_to_buf_index,
                                                 rmm::cuda_stream_view)
  {
    auto start = current;

    if (col.nullable()) {
      std::tie(current, offset_stack_pos) =
        add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
    }

    // info for the data buffer
    *current = src_buf_info(
      col.type().id(), nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
    current++;

    // starts as a count of buffers per input column. will be scanned later.
    metadata_col_to_buf_index.push_back(current - start);

    return {current, offset_stack_pos + offset_depth};
  }

  template <typename T, typename... Args>
  std::enable_if_t<std::is_same_v<T, cudf::dictionary32>, std::pair<src_buf_info*, size_type>>
  operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type");
  }

 private:
  std::pair<src_buf_info*, size_type> add_null_buffer(column_view const& col,
                                                      src_buf_info* current,
                                                      int offset_stack_pos,
                                                      int parent_offset_index,
                                                      int offset_depth)
  {
    // info for the validity buffer
    *current = src_buf_info(
      type_id::INT32, nullptr, offset_stack_pos, parent_offset_index, true, col.offset());

    return {current + 1, offset_stack_pos + offset_depth};
  }
};

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::string_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  std::vector<size_type>& metadata_col_to_buf_index,
  rmm::cuda_stream_view stream)
{
  auto start = current;

  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // the way strings are arranged, the strings column itself contains char data, but our child
  // offsets column actually contains our offsets. So our parent_offset_index is actually our child.

  // string columns don't necessarily have children if they are empty
  auto const has_offsets_child = col.num_children() > 0;

  // string columns contain the underlying chars data.
  *current = src_buf_info(type_id::STRING,
                          nullptr,
                          offset_stack_pos,
                          // if I have an offsets child, it's index will be my parent_offset_index
                          has_offsets_child ? ((current + 1) - head) : parent_offset_index,
                          false,
                          col.offset());

  // if I have offsets, I need to include that in the stack size
  offset_stack_pos += has_offsets_child ? offset_depth + 1 : offset_depth;
  current++;

  if (has_offsets_child) {
    CUDF_EXPECTS(col.num_children() == 1, "Encountered malformed string column");
    strings_column_view scv(col);

    // info for the offsets buffer
    auto offset_col = current;
    CUDF_EXPECTS(not scv.offsets().nullable(), "Encountered nullable string offsets column");
    *current = src_buf_info(type_id::INT32,
                            // note: offsets can be null in the case where the string column
                            // has been created with empty_like().
                            scv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                            offset_stack_pos,
                            parent_offset_index,
                            false,
                            col.offset());

    current++;
    offset_stack_pos += offset_depth;

    // since we are crossing an offset boundary, calculate our new depth and parent offset index.
    offset_depth++;
    parent_offset_index = offset_col - head;
  }

  // starts as a count of buffers per input column. will be scanned later.
  metadata_col_to_buf_index.push_back(current - start);

  return {current, offset_stack_pos};
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::list_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  std::vector<size_type>& metadata_col_to_buf_index,
  rmm::cuda_stream_view stream)
{
  lists_column_view lcv(col);
  auto start = current;

  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // list columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::LIST, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed list column");

  // info for the offsets buffer
  auto offset_col = current;
  *current        = src_buf_info(type_id::INT32,
                          // note: offsets can be null in the case where the lists column
                          // has been created with empty_like().
                          lcv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                          offset_stack_pos,
                          parent_offset_index,
                          false,
                          col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // since we are crossing an offset boundary, calculate our new depth and parent offset index.
  offset_depth++;
  parent_offset_index = offset_col - head;

  // starts as a count of buffers per input column. will be scanned later.
  metadata_col_to_buf_index.push_back(current - start);

  return setup_source_buf_info(col.child_begin() + 1,
                               col.child_end(),
                               head,
                               current,
                               metadata_col_to_buf_index,
                               stream,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::struct_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  std::vector<size_type>& metadata_col_to_buf_index,
  rmm::cuda_stream_view stream)
{
  auto start = current;

  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // struct columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::STRUCT, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // starts as a count of buffers per input column. will be scanned later.
  metadata_col_to_buf_index.push_back(current - start);

  // recurse on children
  cudf::structs_column_view scv(col);
  std::vector<column_view> sliced_children;
  sliced_children.reserve(scv.num_children());
  std::transform(
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(scv.num_children()),
    std::back_inserter(sliced_children),
    [&scv, &stream](size_type child_index) { return scv.get_sliced_child(child_index, stream); });
  return setup_source_buf_info(sliced_children.begin(),
                               sliced_children.end(),
                               head,
                               current,
                               metadata_col_to_buf_index,
                               stream,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          std::vector<size_type>& metadata_col_to_buf_index,
                                                          rmm::cuda_stream_view stream,
                                                          int offset_stack_pos,
                                                          int parent_offset_index,
                                                          int offset_depth)
{
  std::for_each(begin, end, [&](column_view const& col) {
    std::tie(current, offset_stack_pos) = cudf::type_dispatcher(col.type(),
                                                                buf_info_functor{head},
                                                                col,
                                                                current,
                                                                offset_stack_pos,
                                                                parent_offset_index,
                                                                offset_depth,
                                                                metadata_col_to_buf_index,
                                                                stream);
  });
  return {current, offset_stack_pos};
}

/**
 * @brief Given a column, processed split buffers, and a metadata builder, populate
 * the metadata for this column in the builder, and return a tuple of:
 * column size, data offset, bitmask offset and null count.
 *
 * @param src column_view to create metadata from
 * @param current_info dst_buf_info pointer reference, pointing to this column's buffer info
 *                     This is a pointer reference because it is updated by this function as the
 *                     columns's validity and data buffers are visited
 * @param mb A metadata_builder instance to update with the column's packed metadata
 * @param use_src_null_count True for the chunked_pack case where current_info has invalid null
 *                           count information. The null count should be taken
 *                           from `src` because this case is restricted to a single partition
 *                           (no splits)
 * @returns a std::tuple containing:
 *          column size, data offset, bitmask offset, and null count
 */
/*
template <typename BufInfo>
std::tuple<size_type, int64_t, int64_t, size_type> build_output_column_metadata(
  column_view const& src,
  BufInfo& current_info,
  detail::metadata_builder& mb,
  bool use_src_null_count)
{
  auto [bitmask_offset, null_count] = [&]() {
    if (src.nullable()) {
      // offsets in the existing serialized_column metadata are int64_t
      // that's the reason for the casting in this code.
      int64_t const bitmask_offset =
        current_info->num_elements == 0
          ? -1  // this means that the bitmask buffer pointer should be nullptr
          : static_cast<int64_t>(current_info->dst_offset);

      // use_src_null_count is used for the chunked contig split case, where we have
      // no splits: the null_count is just the source column's null_count
      size_type const null_count = use_src_null_count
                                     ? src.null_count()
                                     : (current_info->num_elements == 0
                                          ? 0
                                          : (current_info->num_rows - current_info->valid_count));

      ++current_info;
      return std::pair(bitmask_offset, null_count);
    }
    return std::pair(static_cast<int64_t>(-1), 0);
  }();

  // size/data pointer for the column
  auto const col_size = [&]() {
    // if I am a string column, I need to use the number of rows from my child offset column. the
    // number of rows in my dst_buf_info struct will be equal to the number of chars, which is
    // incorrect. this is a quirk of how cudf stores strings.
    if (src.type().id() == type_id::STRING) {
      // if I have no children (no offsets), then I must have a row count of 0
      if (src.num_children() == 0) { return 0; }

      // otherwise my actual number of rows will be the num_rows field of the next dst_buf_info
      // struct (our child offsets column)
      return (current_info + 1)->num_rows;
    }

    // otherwise the number of rows is the number of elements
    return static_cast<size_type>(current_info->num_elements);
  }();
  int64_t const data_offset =
    col_size == 0 || src.head() == nullptr ? -1 : static_cast<int64_t>(current_info->dst_offset);

  mb.add_column_info_to_meta(
    src.type(), col_size, null_count, data_offset, bitmask_offset, src.num_children());

  ++current_info;
  return {col_size, data_offset, bitmask_offset, null_count};
}
*/

/**
 * @brief Given a set of input columns and processed split buffers, produce
 * output columns.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param out_begin Output iterator of column views
 * @param base_ptr Pointer to the base address of copied data for the working partition
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
/*
template <typename InputIter, typename BufInfo, typename Output>
BufInfo build_output_columns(InputIter begin,
                             InputIter end,
                             BufInfo info_begin,
                             Output out_begin,
                             uint8_t const* const base_ptr,
                             detail::metadata_builder& mb)
{
  auto current_info = info_begin;
  std::transform(begin, end, out_begin, [&current_info, base_ptr, &mb](column_view const& src) {
    auto [col_size, data_offset, bitmask_offset, null_count] =
      build_output_column_metadata<BufInfo>(src, current_info, mb, false);

    auto const bitmask_ptr =
      base_ptr != nullptr && bitmask_offset != -1
        ? reinterpret_cast<bitmask_type const*>(base_ptr + static_cast<uint64_t>(bitmask_offset))
        : nullptr;

    // size/data pointer for the column
    uint8_t const* data_ptr = base_ptr != nullptr && data_offset != -1
                                ? base_ptr + static_cast<uint64_t>(data_offset)
                                : nullptr;

    // children
    auto children = std::vector<column_view>{};
    children.reserve(src.num_children());

    current_info = build_output_columns(
      src.child_begin(), src.child_end(), current_info, std::back_inserter(children), base_ptr, mb);

    return column_view{
      src.type(), col_size, data_ptr, bitmask_ptr, null_count, 0, std::move(children)};
  });

  return current_info;
}
*/

/**
 * @brief Given a set of input columns, processed split buffers, and a metadata_builder,
 * append column metadata using the builder.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param mb packed column metadata builder
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
/*
template <typename InputIter, typename BufInfo>
BufInfo populate_metadata(InputIter begin,
                          InputIter end,
                          BufInfo info_begin,
                          detail::metadata_builder& mb)
{
  auto current_info = info_begin;
  std::for_each(begin, end, [&current_info, &mb](column_view const& src) {
    build_output_column_metadata<BufInfo>(src, current_info, mb, true);

    // children
    current_info = populate_metadata(src.child_begin(), src.child_end(), current_info, mb);
  });

  return current_info;
}
*/

/**
 * @brief Functor that retrieves the size of a destination buffer
 */
struct buf_size_functor {
  dst_buf_info const* ci;
  size_t num_bufs;
  // std::size_t per_partition_metadata_size;
  std::size_t operator() __device__(int index) 
  {
    return index >= num_bufs ? 0 : ci[index].buf_size;
  }
};

/**
 * @brief Functor that retrieves the split "key" for a given output
 * buffer index.
 *
 * The key is simply the partition index.
 */
struct split_key_functor {
  int const num_src_bufs;
  int operator() __device__(int buf_index) const { return buf_index / num_src_bufs; }
};

/**
 * @brief Output iterator for writing values to the dst_offset field of the
 * dst_buf_info struct
 */
struct dst_offset_output_iterator {
  dst_buf_info* c;
  using value_type        = std::size_t;
  using difference_type   = std::size_t;
  using pointer           = std::size_t*;
  using reference         = std::size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_offset_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  dst_offset_output_iterator& operator++ __host__ __device__()
  {
    c++;
    return *this;
  }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->dst_offset; }
};

/**
 * @brief Output iterator for writing values to the valid_count field of the
 * dst_buf_info struct
 */
struct dst_valid_count_output_iterator {
  dst_buf_info* c;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_valid_count_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  dst_valid_count_output_iterator& operator++ __host__ __device__()
  {
    c++;
    return *this;
  }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->valid_count; }
};

/**
 * @brief Functor for computing size of data elements for a given cudf type.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct size_of_helper {
  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && !std::is_same_v<T, cudf::string_view>, size_t>
    __device__ operator()() const
  {    
    return 0;
  }

  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && std::is_same_v<T, cudf::string_view>, size_t>
    __device__ operator()() const
  {
    return sizeof(cudf::device_storage_type_t<int8_t>);
  }

  template <typename T>
  constexpr std::enable_if_t<is_fixed_width<T>(), size_t> __device__ operator()() const noexcept
  {
    return sizeof(cudf::device_storage_type_t<T>);
  }
};

/**
 * @brief Functor for returning the number of batches an input buffer is being
 * subdivided into during the repartitioning step.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct num_batches_func {
  thrust::pair<std::size_t, std::size_t> const* const batches;
  __device__ std::size_t operator()(size_type i) const { return thrust::get<0>(batches[i]); }
};

/**
 * @brief Get the size in bytes of a batch described by `dst_buf_info`.
 */
struct batch_byte_size_function {
  size_type const num_batches;
  dst_buf_info const* const infos;
  __device__ std::size_t operator()(size_type i) const
  {
    if (i == num_batches) { return 0; }
    auto const& buf = *(infos + i);
    std::size_t const bytes =
      static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);
    return util::round_up_unsafe(bytes, shuffle_split_partition_data_align);
  }
};

/**
 * @brief Get the input buffer index given the output buffer index.
 */
struct out_to_in_index_function {
  size_type const* const batch_offsets;
  int const num_bufs;
  __device__ int operator()(size_type i) const
  {
    int ret = static_cast<size_type>(
             thrust::upper_bound(thrust::seq, batch_offsets, batch_offsets + num_bufs + 1, i) -
             batch_offsets) -
           1;    
    return ret;
  }
};

struct partition_buf_size_func {
  cudf::device_span<size_t const> buf_sizes;
  __device__ size_t operator()(int i)
  {
    return i >= buf_sizes.size() ? 0 : buf_sizes[i];
  }
};

// packed block of memory 1: split indices and src_buf_info structs
struct packed_split_indices_and_src_buf_info {
  packed_split_indices_and_src_buf_info(cudf::table_view const& input,
                                        std::vector<size_type> const& splits,
                                        std::size_t num_partitions,
                                        cudf::size_type num_src_bufs,
                                        cz_metadata_internal const& metadata,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref temp_mr)
    : indices_size(
        cudf::util::round_up_safe((num_partitions + 1) * sizeof(size_type), shuffle_split_partition_data_align)),
      src_buf_info_size(
        cudf::util::round_up_safe(num_src_bufs * sizeof(src_buf_info), shuffle_split_partition_data_align)),
      // host-side
      h_indices_and_source_info(indices_size + src_buf_info_size),
      h_indices{reinterpret_cast<size_type*>(h_indices_and_source_info.data())},
      h_src_buf_info{
        reinterpret_cast<src_buf_info*>(h_indices_and_source_info.data() + indices_size)}
  {
    // compute splits -> indices.
    // these are row numbers per split
    h_indices[0]              = 0;
    h_indices[num_partitions] = input.column(0).size();
    std::copy(splits.begin(), splits.end(), std::next(h_indices));

    // mapping of metadata column to src/dst buffer index, which we will need later on to 
    // pack row counts
    std::vector<size_type> metadata_col_to_buf_index;
    metadata_col_to_buf_index.reserve(num_src_bufs); // worst case

    // setup source buf info
    setup_source_buf_info(input.begin(), input.end(), h_src_buf_info, h_src_buf_info, metadata_col_to_buf_index, stream);

    auto const metadata_size = metadata.global_metadata.col_info.size();
    metadata_col_to_buf_index_size = cudf::util::round_up_safe(metadata_size * sizeof(size_type), shuffle_split_partition_data_align);

    offset_stack_partition_size = compute_offset_stack_size(input.begin(), input.end());
    offset_stack_size           = offset_stack_partition_size * num_partitions * sizeof(size_type);
    
    // device-side
    // gpu-only : stack space needed for nested list offset calculation
    d_indices_and_source_info =
      rmm::device_buffer(metadata_col_to_buf_index_size + indices_size + src_buf_info_size + offset_stack_size, stream, temp_mr);
    uint8_t* base_ptr = reinterpret_cast<uint8_t*>(d_indices_and_source_info.data());
    
  
    d_metadata_col_to_buf_index = reinterpret_cast<size_type*>(base_ptr);

    d_indices      = reinterpret_cast<size_type*>(base_ptr + metadata_col_to_buf_index_size);
    d_src_buf_info = reinterpret_cast<src_buf_info*>(base_ptr + metadata_col_to_buf_index_size + indices_size);
    d_offset_stack =
      reinterpret_cast<size_type*>(base_ptr + metadata_col_to_buf_index_size + indices_size + src_buf_info_size);

    // compute metadata col index -> buf index 
    hipMemcpyAsync(d_metadata_col_to_buf_index, metadata_col_to_buf_index.data(), sizeof(size_type) * metadata_col_to_buf_index.size(), hipMemcpyHostToDevice, stream);
    thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                           d_metadata_col_to_buf_index,
                           d_metadata_col_to_buf_index + metadata_size,
                           d_metadata_col_to_buf_index);
    // print_span(cudf::device_span<size_type const>{d_metadata_col_to_buf_index, metadata_size});

    CUDF_CUDA_TRY(hipMemcpyAsync(
      d_indices, h_indices, indices_size + src_buf_info_size, hipMemcpyDefault, stream.value()));
  }

  size_type const indices_size;
  std::size_t const src_buf_info_size;
  std::size_t offset_stack_size;

  std::vector<uint8_t> h_indices_and_source_info;
  rmm::device_buffer d_indices_and_source_info;

  size_type* const h_indices;
  src_buf_info* const h_src_buf_info;

  // data for shuffle split
  size_t metadata_col_to_buf_index_size;
  size_type* d_metadata_col_to_buf_index;

  int offset_stack_partition_size;
  size_type* d_indices;
  src_buf_info* d_src_buf_info;
  size_type* d_offset_stack;
};

// packed block of memory 2: partition buffer sizes and dst_buf_info structs
struct packed_partition_buf_size_and_dst_buf_info {
  packed_partition_buf_size_and_dst_buf_info(std::size_t num_partitions,
                                             std::size_t num_bufs,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref temp_mr)
    : stream(stream),
      partition_sizes_size{cudf::util::round_up_safe(num_partitions * sizeof(std::size_t), shuffle_split_partition_data_align)},
      dst_buf_info_size{cudf::util::round_up_safe(num_bufs * sizeof(dst_buf_info), shuffle_split_partition_data_align)},
      // host-side
      h_partition_sizes_and_dst_info(partition_sizes_size + dst_buf_info_size),
      // h_buf_sizes{reinterpret_cast<std::size_t*>(h_buf_sizes_and_dst_info.data())},
      h_dst_buf_info{
        reinterpret_cast<dst_buf_info*>(h_partition_sizes_and_dst_info.data() + partition_sizes_size)},
      // device-side
      d_partition_sizes_and_dst_info(partition_sizes_size + dst_buf_info_size, stream, temp_mr),
      d_partition_sizes{reinterpret_cast<std::size_t*>(d_partition_sizes_and_dst_info.data())},
      // destination buffer info
      d_dst_buf_info{reinterpret_cast<dst_buf_info*>(
        static_cast<uint8_t*>(d_partition_sizes_and_dst_info.data()) + partition_sizes_size)}
  {
  }

  /*
  void copy_to_host()
  {    
    // DtoH buf sizes and col info back to the host
    CUDF_CUDA_TRY(hipMemcpyAsync(h_buf_sizes,
                                  d_buf_sizes,
                                  buf_sizes_size + dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));                                  
  }
  */

  rmm::cuda_stream_view const stream;

  // partition sizes and destination info (used in batched copies)
  std::size_t const partition_sizes_size;
  std::size_t const dst_buf_info_size;

  std::vector<uint8_t> h_partition_sizes_and_dst_info;
  //std::size_t* const h_buf_sizes;
  dst_buf_info* const h_dst_buf_info;

  std::size_t h_dst_buf_total_size;

  rmm::device_buffer d_partition_sizes_and_dst_info;
  std::size_t* const d_partition_sizes;     // length: the # of partitions
  dst_buf_info* const d_dst_buf_info;       // length: the # of partitions * number of source buffers
};

// Packed block of memory 3:
// Pointers to source and destination buffers (and stack space on the
// gpu for offset computation)
struct packed_src_and_dst_pointers {
  packed_src_and_dst_pointers(cudf::table_view const& input,
                              std::size_t num_partitions,
                              cudf::size_type num_src_bufs,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref temp_mr)
    : stream(stream),
      src_bufs_size{cudf::util::round_up_safe(num_src_bufs * sizeof(uint8_t*), shuffle_split_partition_data_align)},
      dst_bufs_size{cudf::util::round_up_safe(/*num_partitions*/1 * sizeof(uint8_t*), shuffle_split_partition_data_align)},
      // host-side
      h_src_and_dst_buffers(src_bufs_size + dst_bufs_size),
      h_src_bufs{reinterpret_cast<uint8_t const**>(h_src_and_dst_buffers.data())},
      h_dst_buf{reinterpret_cast<uint8_t**>(h_src_and_dst_buffers.data() + src_bufs_size)},
      // device-side
      d_src_and_dst_buffers{rmm::device_buffer(src_bufs_size + dst_bufs_size, stream, temp_mr)},
      d_src_bufs{reinterpret_cast<uint8_t const**>(d_src_and_dst_buffers.data())},
      d_dst_buf{reinterpret_cast<uint8_t**>(
        reinterpret_cast<uint8_t*>(d_src_and_dst_buffers.data()) + src_bufs_size)}
  {
    // setup src buffers
    setup_src_buf_data(input.begin(), input.end(), h_src_bufs);
  }

  void copy_to_device()
  {
    CUDF_CUDA_TRY(hipMemcpyAsync(d_src_and_dst_buffers.data(),
                                  h_src_and_dst_buffers.data(),
                                  src_bufs_size + dst_bufs_size,
                                  hipMemcpyDefault,
                                  stream.value()));
  }

  rmm::cuda_stream_view const stream;
  std::size_t const src_bufs_size;
  std::size_t const dst_bufs_size;

  std::vector<uint8_t> h_src_and_dst_buffers;
  uint8_t const** const h_src_bufs;
  //uint8_t** const h_dst_bufs;
  uint8_t** const h_dst_buf;

  rmm::device_buffer d_src_and_dst_buffers;
  uint8_t const** const d_src_bufs;
  //uint8_t** const d_dst_bufs;
  uint8_t** const d_dst_buf;
};

/**
 * @brief Create an instance of `packed_src_and_dst_pointers` populating destination
 * partition buffers (if any) from `out_buffers`. In the chunked_pack case
 * `out_buffers` is empty, and the destination pointer is provided separately
 * to the `copy_partitions` kernel.
 *
 * @param input source table view
 * @param num_partitions the number of partitions (1 meaning no splits)
 * @param num_src_bufs number of buffers for the source columns including children
 * @param out_buffers the destination buffers per partition if in the non-chunked case
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to packed_src_and_dst_pointers
 */
std::unique_ptr<packed_src_and_dst_pointers> setup_src_and_dst_pointers(
  cudf::table_view const& input,
  std::size_t num_partitions,
  cudf::size_type num_src_bufs,
  rmm::device_buffer& out_buffer,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref temp_mr)
{
  auto src_and_dst_pointers = std::make_unique<packed_src_and_dst_pointers>(
    input, num_partitions, num_src_bufs, stream, temp_mr);

  /*
  std::transform(
    out_buffers.begin(), out_buffers.end(), src_and_dst_pointers->h_dst_bufs, [](auto& buf) {
      return static_cast<uint8_t*>(buf.data());
    });
    */
  src_and_dst_pointers->h_dst_buf[0] = static_cast<uint8_t*>(out_buffer.data());
  

  // copy the struct to device memory to access from the kernel
  src_and_dst_pointers->copy_to_device();

  return src_and_dst_pointers;
}

template <typename InputIter>
std::pair<size_type, size_type> count_internal_columns(InputIter begin, InputIter end, int depth = 0)
{ 
  /*
  auto child_count = [&](column_view const& col){
    if(col.type().id() == cudf::type_id::STRUCT){
      return count_internal_columns(col.child_begin(), col.child_end(), depth+1);
    } else if(col.type().id() == cudf::type_id::LIST){
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      return count_internal_columns(children.begin(), children.end(), depth+1);
    }
    return {0};
  };
  auto buf_iter = thrust::make_transform_iterator(begin, [&](column_view const& col) {
    auto const children = child_count(col);
    return {1 + children.first, 1 + children.second};
  });
  
  return std::accumulate(buf_iter, buf_iter + std::distance(begin, end), {0, 0}, [](std::pair<size_type, size_type> const& a, std::pair<size_type, size_type> const& b) -> std::pair<size_type, size_type>{
    return {a.first + b.first, std::max(a.second, b.second)};
  });
  */

  auto child_count = [&](column_view const& col, int depth) -> std::pair<size_type, size_type> {
    if(col.type().id() == cudf::type_id::STRUCT){
      return count_internal_columns(col.child_begin(), col.child_end(), depth+1);
    } else if(col.type().id() == cudf::type_id::LIST){
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      return count_internal_columns(children.begin(), children.end(), depth+1);
    }
    return {0, depth};
  };

  size_type col_count = 0;
  size_type max_depth = 0;
  std::for_each(begin, end, [&](column_view const& col){
    auto const cc = child_count(col, depth);
    col_count += (1 + cc.first);
    max_depth = std::max(max_depth, cc.second);
  });

  return {col_count, max_depth};
}

template <typename InputIter>
void populate_column_data(cz_metadata_internal& meta, InputIter begin, InputIter end)
{
  std::for_each(begin, end, [&meta](column_view const& col){
    // strings need to store an additional char count
    meta.per_partition_metadata_size += col.type().id() == cudf::type_id::STRING ? 4 : 0;

    switch(col.type().id()){
    case cudf::type_id::STRUCT:
      meta.global_metadata.col_info.push_back({col.type().id(), col.num_children()});
      populate_column_data(meta, col.child_begin(), col.child_end());
      break;
    
    case cudf::type_id::LIST: {
      meta.global_metadata.col_info.push_back({col.type().id(), 1});
      cudf::lists_column_view lcv(col);
      std::vector<cudf::column_view> children({lcv.child()});
      populate_column_data(meta, children.begin(), children.end());
      } break;

    case cudf::type_id::DECIMAL32:
    case cudf::type_id::DECIMAL64:
    case cudf::type_id::DECIMAL128:
      // TODO: scale.
      meta.global_metadata.col_info.push_back({col.type().id(), 0});
      break;

    default:
      meta.global_metadata.col_info.push_back({col.type().id(), 0});
      break;
    }
  });
}

// returns global metadata describing the table and the size of the
// internal per-partition data
cz_metadata_internal compute_metadata(cudf::table_view const& input)
{
  auto const [num_internal_columns, max_depth] = count_internal_columns(input.begin(), input.end());

  // compute the metadata
  cz_metadata_internal ret;
  ret.global_metadata.col_info.reserve(num_internal_columns);
  // 4 byte row count
  ret.per_partition_metadata_size += 4;
  // 1 bit indicating presence of null vector, per internal column
  ret.per_partition_metadata_size += (cudf::util::round_up_safe(num_internal_columns, 32) / 32) * sizeof(bitmask_type);
  populate_column_data(ret, input.begin(), input.end());
  // pad out to shuffle_split_partition_data_align bytes
  ret.per_partition_metadata_size = cudf::util::round_up_safe(ret.per_partition_metadata_size, static_cast<size_type>(shuffle_split_partition_data_align));
  ret.max_depth = max_depth;

  return ret;
}

/**
 * @brief Create an instance of `packed_partition_buf_size_and_dst_buf_info` containing
 * the partition-level dst_buf_info structs for each partition and column buffer.
 *
 * @param input source table view
 * @param splits the numeric value (in rows) for each split, empty for 1 partition
 * @param num_partitions the number of partitions create (1 meaning no splits)
 * @param num_src_bufs number of buffers for the source columns including children
 * @param num_bufs num_src_bufs times the number of partitions
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to `packed_partition_buf_size_and_dst_buf_info`
 */
std::pair<std::unique_ptr<packed_partition_buf_size_and_dst_buf_info>, std::unique_ptr<packed_split_indices_and_src_buf_info>> compute_splits(
  cudf::table_view const& input,
  std::vector<size_type> const& splits,
  std::size_t num_partitions,
  cudf::size_type num_src_bufs,
  std::size_t num_bufs,
  cz_metadata_internal const& metadata,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref temp_mr)
{
  auto partition_buf_size_and_dst_buf_info =
    std::make_unique<packed_partition_buf_size_and_dst_buf_info>(
      num_partitions, num_bufs, stream, temp_mr);

  auto const d_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
  auto const d_partition_sizes    = partition_buf_size_and_dst_buf_info->d_partition_sizes;

  auto split_indices_and_src_buf_info = std::make_unique<packed_split_indices_and_src_buf_info>(
    input, splits, num_partitions, num_src_bufs, metadata, stream, temp_mr);

  auto const d_src_buf_info = split_indices_and_src_buf_info->d_src_buf_info;
  auto const offset_stack_partition_size =
    split_indices_and_src_buf_info->offset_stack_partition_size;
  auto const d_offset_stack = split_indices_and_src_buf_info->d_offset_stack;
  auto const d_indices      = split_indices_and_src_buf_info->d_indices;

  // compute sizes of each column in each partition, including alignment.
  thrust::transform(
    rmm::exec_policy_nosync(stream, temp_mr),
    thrust::make_counting_iterator<std::size_t>(0),
    thrust::make_counting_iterator<std::size_t>(num_bufs),
    d_dst_buf_info,
    cuda::proclaim_return_type<dst_buf_info>([d_src_buf_info,
                                              offset_stack_partition_size,
                                              d_offset_stack,
                                              d_indices,
                                              num_src_bufs] __device__(std::size_t t) {
      int const split_index   = t / num_src_bufs;
      int const src_buf_index = t % num_src_bufs;
      auto const& src_info    = d_src_buf_info[src_buf_index];

      // apply nested offsets (lists and string columns).
      //
      // We can't just use the incoming row indices to figure out where to read from in a
      // nested list situation.  We have to apply offsets every time we cross a boundary
      // (list or string).  This loop applies those offsets so that our incoming row_index_start
      // and row_index_end get transformed to our final values.
      //
      int const stack_pos = src_info.offset_stack_pos + (split_index * offset_stack_partition_size);
      size_type* offset_stack  = &d_offset_stack[stack_pos];
      int parent_offsets_index = src_info.parent_offsets_index;
      int stack_size           = 0;
      int root_column_offset   = src_info.column_offset;
      int const root_row_start = d_indices[split_index] + root_column_offset;
      int const root_row_end = d_indices[split_index + 1] + root_column_offset;
      int const root_row_count = root_row_end - root_row_start;
      while (parent_offsets_index >= 0) {
        offset_stack[stack_size++] = parent_offsets_index;
        root_column_offset         = d_src_buf_info[parent_offsets_index].column_offset;
        parent_offsets_index       = d_src_buf_info[parent_offsets_index].parent_offsets_index;
      }
      // make sure to include the -column- offset on the root column in our calculation.
      int row_start = d_indices[split_index] + root_column_offset;
      int row_end   = d_indices[split_index + 1] + root_column_offset;
      while (stack_size > 0) {
        stack_size--;
        auto const offsets = d_src_buf_info[offset_stack[stack_size]].offsets;
        // this case can happen when you have empty string or list columns constructed with
        // empty_like()
        if (offsets != nullptr) {
          row_start = offsets[row_start];
          row_end   = offsets[row_end];
        }
      }

      // final element indices and row count
      int const src_element_index = src_info.is_validity ? row_start / 32 : row_start;
      int const num_rows          = row_end - row_start;
      // if I am an offsets column, all my values need to be shifted
      int const value_shift = src_info.offsets == nullptr ? 0 : src_info.offsets[row_start];
      // if I am a validity column, we may need to shift bits
      int const bit_shift = src_info.is_validity ? row_start % 32 : 0;
      // # of rows isn't necessarily the same as # of elements to be copied.
      auto const num_elements = [&]() {
        if (src_info.offsets != nullptr && num_rows > 0) {
          return num_rows + 1;
        } else if (src_info.is_validity) {
          return (num_rows + 31) / 32;
        }
        return num_rows;
      }();
      int const element_size = cudf::type_dispatcher(data_type{src_info.type}, size_of_helper{});
      std::size_t const bytes =
        static_cast<std::size_t>(num_elements) * static_cast<std::size_t>(element_size);
      return dst_buf_info{util::round_up_unsafe(bytes, shuffle_split_partition_data_align),
                          num_elements,
                          element_size,
                          num_rows,
                          src_element_index,
                          0,
                          value_shift,
                          bit_shift,
                          src_info.is_validity ? 1 : 0,
                          src_buf_index,
                          root_row_count};
    }));
  
  // - compute total size of each partition and total buffer size overall
  // - compute start offset for each destination buffer within each split  
  {
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, split_key_functor{static_cast<int>(num_src_bufs)});
    auto buf_sizes =
      cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info, num_bufs});

    // reduce to compute sizes, then add in per_partition_metadata_size
    thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                          keys,
                          keys + num_bufs,
                          buf_sizes,
                          thrust::make_discard_iterator(),
                          d_partition_sizes);
    
    /*
    {
      std::vector<size_t> h_partition_sizes(num_partitions);
      hipMemcpy(h_partition_sizes.data(), d_partition_sizes, sizeof(size_t) * num_partitions, hipMemcpyDeviceToHost);
      for(size_t idx=0; idx<num_partitions; idx++){
        printf("HBS(%lu): %lu\n", idx, h_partition_sizes[idx]);
      }
    }
    */
    thrust::transform(rmm::exec_policy_nosync(stream, temp_mr),
                      d_partition_sizes,
                      d_partition_sizes + num_partitions,
                      d_partition_sizes,
                      [per_partition_metadata_size = metadata.per_partition_metadata_size] __device__ (std::size_t partition_size){
                        return util::round_up_unsafe(partition_size + per_partition_metadata_size, shuffle_split_partition_data_align);
                      });

    // print_span(cudf::device_span<size_t const>{d_partition_sizes, num_partitions});
    
    // total size
    partition_buf_size_and_dst_buf_info->h_dst_buf_total_size = thrust::reduce(rmm::exec_policy(stream, temp_mr),
                                                                               d_partition_sizes,
                                                                               d_partition_sizes + num_partitions);

    // scan to per-partition destination buf offsets (num_src_bufs * num_partitions), then add metdata offset
    thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr),
                           buf_sizes,
                           buf_sizes + num_bufs,
                           dst_offset_output_iterator{d_dst_buf_info},
                           std::size_t{0});
    // add metadata header offset
    auto iter = thrust::make_counting_iterator(0);
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + num_bufs,
                    [per_partition_metadata_size = metadata.per_partition_metadata_size,
                     bufs_per_partition = num_src_bufs,
                     d_dst_buf_info]  __device__ (size_type i){

      auto const partition_index = i / bufs_per_partition;
      auto const metadata_offset = (partition_index + 1) * per_partition_metadata_size;
      d_dst_buf_info[i].dst_offset += metadata_offset;
      // printf("dst(%i): %lu\n", i, d_dst_buf_info[i].dst_offset);
    });    
  }

  /*
  // compute start offset for each output buffer for each split
  {
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, split_key_functor{static_cast<int>(num_src_bufs)});
    auto values =
      cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info});

    thrust::exclusive_scan_by_key(rmm::exec_policy(stream, temp_mr),
                                  keys,
                                  keys + num_bufs,
                                  values,
                                  dst_offset_output_iterator{d_dst_buf_info},
                                  std::size_t{0});
  }
  */

  // compute start offset for each destination buffer within each split  
  {
    /*
    auto sizes =
        cudf::detail::make_counting_transform_iterator(0, [num_bufs, d_buf_sizes] __device__ (size_t i) -> size_t {
          return i >= num_bufs ? 0 : d_buf_sizes[i];
        });
        */
       /*
    thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr),
                           d_buf_sizes,
                           d_buf_sizes + num_bufs,
                           dst_offset_output_iterator{d_dst_buf_info},
                           std::size_t{0});
                           */
    
    /*
    size_t last_offset;
    size_t last_size;
    hipMemcpyAsync(&last_offset, &(d_dst_buf_info[num_bufs-1].dst_offset), sizeof(size_t), hipMemcpyDeviceToHost, stream);    
    hipMemcpyAsync(&last_size, &d_buf_sizes[num_bufs-1], sizeof(size_t), hipMemcpyDeviceToHost, stream);
    stream.synchronize();
    partition_buf_size_and_dst_buf_info->h_dst_buf_total_size = last_offset + last_size;
    */
    
    // blech
    // hipMemcpyAsync(&(partition_buf_size_and_dst_buf_info->h_dst_buf_total_size), &(d_dst_buf_info[num_bufs].dst_offset), sizeof(size_t), hipMemcpyDeviceToHost, stream);
  }

  // partition_buf_size_and_dst_buf_info->copy_to_host();

  return {std::move(partition_buf_size_and_dst_buf_info), std::move(split_indices_and_src_buf_info)};
}

/**
 * @brief Struct containing information about the actual batches we will send to the
 * `copy_partitions` kernel and the number of iterations we need to carry out this copy.
 *
 * For the non-chunked contiguous_split case, this contains the batched dst_buf_infos and the
 * number of iterations is going to be 1 since the non-chunked case is single pass.
 *
 * For the chunked_pack case, this also contains the batched dst_buf_infos for all
 * iterations in addition to helping keep the state about what batches have been copied so far
 * and what are the sizes (in bytes) of each iteration.
 */
struct chunk_iteration_state {
  chunk_iteration_state(rmm::device_uvector<dst_buf_info> _d_batched_dst_buf_info,
                        rmm::device_uvector<size_type> _d_batch_offsets,
                        std::vector<std::size_t>&& _h_num_buffs_per_iteration,
                        std::vector<std::size_t>&& _h_size_of_buffs_per_iteration,
                        std::size_t total_size)
    : num_iterations(_h_num_buffs_per_iteration.size()),
      current_iteration{0},
      starting_batch{0},
      d_batched_dst_buf_info(std::move(_d_batched_dst_buf_info)),
      d_batch_offsets(std::move(_d_batch_offsets)),
      h_num_buffs_per_iteration(std::move(_h_num_buffs_per_iteration)),
      h_size_of_buffs_per_iteration(std::move(_h_size_of_buffs_per_iteration)),
      total_size(total_size)
  {
  }

  static std::unique_ptr<chunk_iteration_state> create(
    rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> const& batches,
    int num_bufs,
    dst_buf_info* d_orig_dst_buf_info,
    std::size_t h_dst_buf_total_size,
    std::size_t num_partitions,
    std::size_t user_buffer_size,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref temp_mr);

  /**
   * @brief As of the time of the call, return the starting 1MB batch index, and the
   * number of batches to copy.
   *
   * @return the current iteration's starting_batch and batch count as a pair
   */
  std::pair<std::size_t, std::size_t> get_current_starting_index_and_buff_count() const
  {
    CUDF_EXPECTS(current_iteration < num_iterations,
                 "current_iteration cannot exceed num_iterations");
    auto count_for_current = h_num_buffs_per_iteration[current_iteration];
    return {starting_batch, count_for_current};
  }

  /**
   * @brief Advance the iteration state if there are iterations left, updating the
   * starting batch and returning the amount of bytes were copied in the iteration
   * we just finished.
   * @throws cudf::logic_error If the state was at the last iteration before entering
   * this function.
   * @return size in bytes that were copied in the finished iteration
   */
  std::size_t advance_iteration()
  {
    CUDF_EXPECTS(current_iteration < num_iterations,
                 "current_iteration cannot exceed num_iterations");
    std::size_t bytes_copied = h_size_of_buffs_per_iteration[current_iteration];
    starting_batch += h_num_buffs_per_iteration[current_iteration];
    ++current_iteration;
    return bytes_copied;
  }

  /**
   * Returns true if there are iterations left.
   */
  bool has_more_copies() const { return current_iteration < num_iterations; }

  rmm::device_uvector<dst_buf_info> d_batched_dst_buf_info;  ///< dst_buf_info per 1MB batch
  rmm::device_uvector<size_type> const d_batch_offsets;  ///< Offset within a batch per dst_buf_info
  std::size_t const total_size;                          ///< The aggregate size of all iterations
  int const num_iterations;                              ///< The total number of iterations
  int current_iteration;  ///< Marks the current iteration being worked on

 private:
  std::size_t starting_batch;  ///< Starting batch index for the current iteration
  std::vector<std::size_t> const h_num_buffs_per_iteration;  ///< The count of batches per iteration
  std::vector<std::size_t> const
    h_size_of_buffs_per_iteration;  ///< The size in bytes per iteration
};

std::unique_ptr<chunk_iteration_state> chunk_iteration_state::create(
  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> const& batches,
  int num_bufs,
  dst_buf_info* d_orig_dst_buf_info,
  std::size_t h_dst_buf_total_size,
  std::size_t num_partitions,
  std::size_t user_buffer_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref temp_mr)
{
  rmm::device_uvector<size_type> d_batch_offsets(num_bufs + 1, stream, temp_mr);

  auto const buf_count_iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<std::size_t>(
      [num_bufs, num_batches = num_batches_func{batches.begin()}] __device__(size_type i) {
        return i == num_bufs ? 0 : num_batches(i);
      }));

  thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                         buf_count_iter,
                         buf_count_iter + num_bufs + 1,
                         d_batch_offsets.begin(),
                         0);

  auto const num_batches_iter =
    cudf::detail::make_counting_transform_iterator(0, num_batches_func{batches.begin()});
  size_type const num_batches = thrust::reduce(
    rmm::exec_policy(stream, temp_mr), num_batches_iter, num_batches_iter + batches.size());

  auto out_to_in_index = out_to_in_index_function{d_batch_offsets.begin(), num_bufs};

  auto const iter = thrust::make_counting_iterator(0);

  // load up the batches as d_dst_buf_info
  rmm::device_uvector<dst_buf_info> d_batched_dst_buf_info(num_batches, stream, temp_mr);

  thrust::for_each(
    rmm::exec_policy(stream, temp_mr),
    iter,
    iter + num_batches,
    [d_orig_dst_buf_info,
     d_batched_dst_buf_info = d_batched_dst_buf_info.begin(),
     batches                = batches.begin(),
     d_batch_offsets        = d_batch_offsets.begin(),
     out_to_in_index] __device__(size_type i) {
      size_type const in_buf_index = out_to_in_index(i);
      size_type const batch_index  = i - d_batch_offsets[in_buf_index];
      auto const batch_size        = thrust::get<1>(batches[in_buf_index]);
      dst_buf_info const& in       = d_orig_dst_buf_info[in_buf_index];

      // adjust info
      dst_buf_info& out = d_batched_dst_buf_info[i];
      out.element_size  = in.element_size;
      out.value_shift   = in.value_shift;
      out.bit_shift     = in.bit_shift;
      out.valid_count =
        in.valid_count;  // valid count will be set to 1 if this is a validity buffer
      out.src_buf_index = in.src_buf_index;
      // out.dst_buf_index = in.dst_buf_index;

      size_type const elements_per_batch =
        out.element_size == 0 ? 0 : batch_size / out.element_size;
      out.num_elements = ((batch_index + 1) * elements_per_batch) > in.num_elements
                           ? in.num_elements - (batch_index * elements_per_batch)
                           : elements_per_batch;

      size_type const rows_per_batch =
        // if this is a validity buffer, each element is a bitmask_type, which
        // corresponds to 32 rows.
        out.valid_count > 0
          ? elements_per_batch * static_cast<size_type>(cudf::detail::size_in_bits<bitmask_type>())
          : elements_per_batch;
      out.num_rows = ((batch_index + 1) * rows_per_batch) > in.num_rows
                       ? in.num_rows - (batch_index * rows_per_batch)
                       : rows_per_batch;

      out.src_element_index = in.src_element_index + (batch_index * elements_per_batch);
      
      out.dst_offset        = in.dst_offset + (batch_index * batch_size);
      // printf("IDO: %lu %d %d\n", in.dst_offset, (int)batch_index, (int)batch_size);

      // out.bytes and out.buf_size are unneeded here because they are only used to
      // calculate real output buffer sizes. the data we are generating here is
      // purely intermediate for the purposes of doing more uniform copying of data
      // underneath the final structure of the output
    });

  /**
   * In the chunked case, this is the code that fixes up the offsets of each batch
   * and prepares each iteration. Given the batches computed before, it figures
   * out the number of batches that will fit in an iteration of `user_buffer_size`.
   *
   * Specifically, offsets for batches are reset to the 0th byte when a new iteration
   * of `user_buffer_size` bytes is needed.
   */
  if (user_buffer_size != 0) {
    // copy the batch offsets back to host
    std::vector<std::size_t> h_offsets(num_batches + 1);
    {
      rmm::device_uvector<std::size_t> offsets(h_offsets.size(), stream, temp_mr);
      auto const batch_byte_size_iter = cudf::detail::make_counting_transform_iterator(
        0, batch_byte_size_function{num_batches, d_batched_dst_buf_info.begin()});

      thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                             batch_byte_size_iter,
                             batch_byte_size_iter + num_batches + 1,
                             offsets.begin());

      CUDF_CUDA_TRY(hipMemcpyAsync(h_offsets.data(),
                                    offsets.data(),
                                    sizeof(std::size_t) * offsets.size(),
                                    hipMemcpyDefault,
                                    stream.value()));

      // the next part is working on the CPU, so we want to synchronize here
      stream.synchronize();
    }

    std::vector<std::size_t> num_batches_per_iteration;
    std::vector<std::size_t> size_of_batches_per_iteration;
    std::vector<std::size_t> accum_size_per_iteration;
    std::size_t accum_size = 0;
    {
      auto current_offset_it = h_offsets.begin();
      // figure out how many iterations we need, while fitting batches to iterations
      // with no more than user_buffer_size bytes worth of batches
      while (current_offset_it != h_offsets.end()) {
        // next_iteration_it points to the batch right above the boundary (the batch
        // that didn't fit).
        auto next_iteration_it =
          std::lower_bound(current_offset_it,
                           h_offsets.end(),
                           // We add the cumulative size + 1 because we want to find what would fit
                           // within a buffer of user_buffer_size (up to user_buffer_size).
                           // Since h_offsets is a prefix scan, we add the size we accumulated so
                           // far so we are looking for the next user_buffer_sized boundary.
                           user_buffer_size + accum_size + 1);

        // we subtract 1 from the number of batch here because next_iteration_it points
        // to the batch that didn't fit, so it's one off.
        auto batches_in_iter = std::distance(current_offset_it, next_iteration_it) - 1;

        // to get the amount of bytes in this iteration we get the prefix scan size
        // and subtract the cumulative size so far, leaving the bytes belonging to this
        // iteration
        auto iter_size_bytes = *(current_offset_it + batches_in_iter) - accum_size;
        accum_size += iter_size_bytes;

        num_batches_per_iteration.push_back(batches_in_iter);
        size_of_batches_per_iteration.push_back(iter_size_bytes);
        accum_size_per_iteration.push_back(accum_size);

        if (next_iteration_it == h_offsets.end()) { break; }

        current_offset_it += batches_in_iter;
      }
    }

    // apply changed offset
    {
      auto d_accum_size_per_iteration =
        cudf::detail::make_device_uvector_async(accum_size_per_iteration, stream, temp_mr);

      // we want to update the offset of batches for every iteration, except the first one (because
      // offsets in the first iteration are all 0 based)
      auto num_batches_in_first_iteration = num_batches_per_iteration[0];
      auto const iter     = thrust::make_counting_iterator(num_batches_in_first_iteration);
      auto num_iterations = accum_size_per_iteration.size();
      thrust::for_each(
        rmm::exec_policy(stream, temp_mr),
        iter,
        iter + num_batches - num_batches_in_first_iteration,
        [num_iterations,
         d_batched_dst_buf_info     = d_batched_dst_buf_info.begin(),
         d_accum_size_per_iteration = d_accum_size_per_iteration.begin()] __device__(size_type i) {
          auto prior_iteration_size =
            thrust::upper_bound(thrust::seq,
                                d_accum_size_per_iteration,
                                d_accum_size_per_iteration + num_iterations,
                                d_batched_dst_buf_info[i].dst_offset) -
            1;
          d_batched_dst_buf_info[i].dst_offset -= *prior_iteration_size;
        });
    }
    return std::make_unique<chunk_iteration_state>(std::move(d_batched_dst_buf_info),
                                                   std::move(d_batch_offsets),
                                                   std::move(num_batches_per_iteration),
                                                   std::move(size_of_batches_per_iteration),
                                                   accum_size);

  } else {
    // we instantiate an "iteration state" for the regular single pass contiguous_split
    // consisting of 1 iteration with all of the batches and totalling `total_size` bytes.
    // auto const total_size = std::reduce(h_buf_sizes, h_buf_sizes + num_partitions);
    auto const total_size = h_dst_buf_total_size;

    // 1 iteration with the whole size
    return std::make_unique<chunk_iteration_state>(
      std::move(d_batched_dst_buf_info),
      std::move(d_batch_offsets),
      std::move(std::vector<std::size_t>{static_cast<std::size_t>(num_batches)}),
      std::move(std::vector<std::size_t>{total_size}),
      total_size);
  }
}

/**
 * @brief Create an instance of `chunk_iteration_state` containing 1MB batches of work
 * that are further grouped into chunks or iterations.
 *
 * This function handles both the `chunked_pack` case: when `user_buffer_size` is non-zero,
 * and the single-shot `contiguous_split` case.
 *
 * @param num_bufs num_src_bufs times the number of partitions
 * @param d_dst_buf_info dst_buf_info per partition produced in `compute_splits`
 * @param h_buf_sizes size in bytes of a partition (accessible from host)
 * @param num_partitions the number of partitions (1 meaning no splits)
 * @param user_buffer_size if non-zero, it is the size in bytes that 1MB batches should be
 *        grouped in, as different iterations.
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to `chunk_iteration_state`
 */
std::unique_ptr<chunk_iteration_state> compute_batches(int num_bufs,
                                                       dst_buf_info* const d_dst_buf_info,
                                                       std::size_t h_dst_buf_total_size,
                                                       std::size_t num_partitions,
                                                       std::size_t user_buffer_size,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::device_async_resource_ref temp_mr)
{
  // Since we parallelize at one block per copy, performance is vulnerable to situations where we
  // have small numbers of copies to do (a combination of small numbers of splits and/or columns),
  // so we will take the actual set of outgoing source/destination buffers and further partition
  // them into much smaller batches in order to drive up the number of blocks and overall
  // occupancy.
  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> batches(num_bufs, stream, temp_mr);
  thrust::transform(
    rmm::exec_policy(stream, temp_mr),
    d_dst_buf_info,
    d_dst_buf_info + num_bufs,
    batches.begin(),
    cuda::proclaim_return_type<thrust::pair<std::size_t, std::size_t>>(
      [desired_batch_size = desired_batch_size] __device__(
        dst_buf_info const& buf) -> thrust::pair<std::size_t, std::size_t> {
        // Total bytes for this incoming partition
        std::size_t const bytes =
          static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);

        // This clause handles nested data types (e.g. list or string) that store no data in the row
        // columns, only in their children.
        if (bytes == 0) { return {1, 0}; }

        // The number of batches we want to subdivide this buffer into
        std::size_t const num_batches = size_to_batch_count(bytes);

        // NOTE: leaving batch size as a separate parameter for future tuning
        // possibilities, even though in the current implementation it will be a
        // constant.
        return {num_batches, desired_batch_size};
      }));

  return chunk_iteration_state::create(batches,
                                       num_bufs,
                                       d_dst_buf_info,
                                       h_dst_buf_total_size,
                                       num_partitions,
                                       user_buffer_size,
                                       stream,
                                       temp_mr);
}

void copy_data(int num_batches_to_copy,
               int starting_batch,
               uint8_t const** d_src_bufs,
               uint8_t** d_dst_buf,
               rmm::device_uvector<dst_buf_info>& d_dst_buf_info,
               uint8_t* user_buffer,
               rmm::cuda_stream_view stream)
{
  constexpr size_type block_size = 256;
  if (user_buffer != nullptr) {
    auto index_to_buffer = [user_buffer] __device__(unsigned int) { return user_buffer; };
    copy_partitions<block_size><<<num_batches_to_copy, block_size, 0, stream.value()>>>(
      index_to_buffer, d_src_bufs, d_dst_buf_info.data() + starting_batch);
  } else {
    // there is only ever 1 destination in the shuffle-split case and all offsets into it are absolute
    auto index_to_buffer = [d_dst_buf/*,
                            dst_buf_info = d_dst_buf_info.data(),
                            user_buffer*/] __device__(unsigned int buf_index) {
      // auto const dst_buf_index = dst_buf_info[buf_index].dst_buf_index;
      return d_dst_buf[0];
    };
    copy_partitions<block_size><<<num_batches_to_copy, block_size, 0, stream.value()>>>(
      index_to_buffer, d_src_bufs, d_dst_buf_info.data() + starting_batch);
  }
}

/**
 * @brief Function that checks an input table_view and splits for specific edge cases.
 *
 * It will return true if the input is "empty" (no rows or columns), which means
 * special handling has to happen in the calling code.
 *
 * @param input table_view of source table to be split
 * @param splits the splits specified by the user, or an empty vector if no splits
 * @returns true if the input is empty, false otherwise
 */
bool check_inputs(cudf::table_view const& input, std::vector<size_type> const& splits)
{
  if (input.num_columns() == 0) { return true; }
  if (splits.size() > 0) {
    CUDF_EXPECTS(splits.back() <= input.column(0).size(),
                 "splits can't exceed size of input columns",
                 std::out_of_range);
  }
  size_type begin = 0;
  for (auto end : splits) {
    CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.", std::out_of_range);
    CUDF_EXPECTS(
      end >= begin, "End index cannot be smaller than the starting index.", std::invalid_argument);
    CUDF_EXPECTS(end <= input.column(0).size(), "Slice range out of bounds.", std::out_of_range);
    begin = end;
  }
  return input.column(0).size() == 0;
}

constexpr size_type type_to_additional_row_counts(cudf::type_id type)
{
  return type == cudf::type_id::STRING ? 1 : 0;
}

__global__ void pack_per_partition_data_kernel(uint8_t* out_buffer,
                                               size_type num_partitions,
                                               size_t columns_per_partition,
                                               src_buf_info const* src_buf_info,
                                               dst_buf_info const* dst_buf_info,
                                               size_type bufs_per_partition,
                                               size_type const* metadata_col_to_buf_index,
                                               size_t const* out_buffer_offsets,
                                               size_type const *char_count_offsets,
                                               shuffle_split_col_data const* col_data)

{
  int const tid = threadIdx.x + blockIdx.x * blockDim.x;
  auto const threads_per_partition = cudf::util::round_up_safe(columns_per_partition, static_cast<size_t>(cudf::detail::warp_size));
  auto const partition_index = tid / threads_per_partition;
  if(partition_index >= num_partitions){
    return;
  }  
  auto const col_index = tid % threads_per_partition;

  // start of the metadata buffer for this partition
  uint8_t* buf_start = out_buffer + out_buffer_offsets[partition_index];

  // first thread in each partition stores the partition-level row count
  if(col_index == 0){
    size_type partition_num_rows = 0;
    // it is possible to get in here with no columns -or- no rows.
    if(col_index < columns_per_partition){
      auto const src_buf_index = metadata_col_to_buf_index[col_index];      
      auto const dst_buf_index = (partition_index * bufs_per_partition) + src_buf_index;
      partition_num_rows = col_data[col_index].type == cudf::type_id::STRING ? dst_buf_info[dst_buf_index].root_num_rows : dst_buf_info[dst_buf_index].num_rows;
      // printf("CBI: %d %d %d %d\n", (int)col_index, (int)src_buf_index, (int)dst_buf_index, (int)partition_num_rows);
    }
    reinterpret_cast<size_type*>(buf_start)[0] = partition_num_rows;
  }  

  // store char count for strings
  if(col_index < columns_per_partition && col_data[col_index].type == cudf::type_id::STRING){
    auto const src_buf_index = metadata_col_to_buf_index[col_index];
    auto const dst_buf_index = (partition_index * bufs_per_partition) + src_buf_index;

    // char count for this column
    size_type* char_count = reinterpret_cast<size_type*>(buf_start + (char_count_offsets[col_index] * sizeof(size_type)) + 4);
    char_count[0] = dst_buf_info[dst_buf_index].num_rows;      // # of chars
  }  

  // store has-validity bits
  bitmask_type mask = __ballot_sync(0xffffffff, col_index < columns_per_partition ? src_buf_info[metadata_col_to_buf_index[col_index]].is_validity : 0);
  if((col_index % cudf::detail::warp_size == 0) && col_index < columns_per_partition){
    auto const num_char_counts = char_count_offsets[columns_per_partition];
    bitmask_type* has_validity = reinterpret_cast<bitmask_type*>(buf_start + (num_char_counts * sizeof(size_type)) + 4);
    // printf("HV: %d : %d, %d, %d\n", (int)(col_index / cudf::detail::warp_size), (int)mask, (int)col_index, (int)tid);
    has_validity[col_index / cudf::detail::warp_size] = mask;
  }
}

// the partition header consists of:
// - an array of size_type elements, representing row counts, corresponding to a column in the global metadata.
//   - string columns contain two row counts (the column row count and the number of chars)
//   - all other columns contain 1 row count
//
// - 1 bit per column in the metadata corresponding to whether or not the column contains validity. rounded up
//   to the nearest byte at the last element
//
// - final padding out to 8 bytes (the minimum alignment needed for the re-assembly step on the receiver side)
//
void pack_per_partition_data(cz_metadata_internal const& metadata,
                             rmm::device_buffer& out_buffer,
                             rmm::device_uvector<size_t> const& out_buffer_offsets,
                             src_buf_info const* d_src_buf_info,
                             dst_buf_info const* d_dst_buf_info,
                             int bufs_per_partition,
                             size_type const* d_metadata_col_to_buf_index,
                             rmm::cuda_stream_view stream)
{
  auto temp_mr = cudf::get_current_device_resource_ref();

  auto const metadata_size = metadata.global_metadata.col_info.size();

  // compute offset for each char count for each string column in the input
  rmm::device_uvector<shuffle_split_col_data> d_col_data = cudf::detail::make_device_uvector_async(metadata.global_metadata.col_info, stream, temp_mr);
  rmm::device_uvector<size_type> char_count_offsets(metadata_size + 1, stream, temp_mr);
  auto char_count_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([d_col_data = d_col_data.begin(), num_cols = d_col_data.size()] __device__ (size_type i) -> size_type {
    return i >= num_cols ? 0 : (d_col_data[i].type == cudf::type_id::STRING ? 1 : 0);
  }));
  thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr),
                         char_count_iter,
                         char_count_iter + metadata_size + 1,
                         char_count_offsets.begin());
  // print_vector(char_count_offsets);
  
  // pack the row counts and validity info
  auto const num_partitions = out_buffer_offsets.size();
  
  // we want a multiple of full warps per partition
  size_type const thread_count_per_partition = cudf::util::round_up_safe(metadata_size, static_cast<size_t>(cudf::detail::warp_size));
  cudf::detail::grid_1d const grid{thread_count_per_partition * static_cast<size_type>(num_partitions), 128};
  pack_per_partition_data_kernel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(reinterpret_cast<uint8_t*>(out_buffer.data()),
                                                                                                     num_partitions,
                                                                                                     metadata_size,
                                                                                                     d_src_buf_info,
                                                                                                     d_dst_buf_info,
                                                                                                     bufs_per_partition,
                                                                                                     d_metadata_col_to_buf_index,
                                                                                                     out_buffer_offsets.begin(),
                                                                                                     char_count_offsets.begin(),
                                                                                                     d_col_data.begin());

  /*
  {
    stream.synchronize();
    std::vector<uint8_t> h_partitions = cudf::detail::make_std_vector_sync(cudf::device_span<uint8_t const>{reinterpret_cast<uint8_t const*>(out_buffer.data()), out_buffer.size()}, stream);
    std::vector<size_t> h_partition_offsets = cudf::detail::make_std_vector_sync(out_buffer_offsets, stream);
    std::vector<size_type> h_row_count_offsets = cudf::detail::make_std_vector_sync(row_count_offsets, stream);

    for(int p_idx=0; p_idx<num_partitions; p_idx++){
      size_type const* row_counts = reinterpret_cast<size_type const*>(h_partitions.data() + h_partition_offsets[p_idx]);
      bitmask_type const* has_validity = reinterpret_cast<bitmask_type const*>(row_counts + row_count_offsets[metadata_size]);
      for(int idx=0; idx<metadata_size; idx++){
      }
    }
  }
  */

#if 0
  // store the row counts and has-validity
  auto iter = thrust::make_counting_iterator(0);
  auto const num_partitions = out_buffer_offsets.size();
  thrust::for_each(rmm::exec_policy_nosync(stream, temp_mr),
                   iter,
                   iter + (metadata_size * num_partitions),
                   [num_partitions,
                    metadata_size,
                    d_src_buf_info,
                    d_dst_buf_info,
                    src_bufs_per_partition,
                    d_metadata_col_to_buf_index,
                    out_buffer = reinterpret_cast<uint8_t*>(out_buffer.data()),
                    out_buffer_offsets = out_buffer_offsets.begin(),
                    row_count_offsets = row_count_offsets.begin(),
                    d_col_data = d_col_data.begin()] __device__ (size_type i){

    auto const partition_index = i / num_partitions;
    auto const col_index = i % num_partitions;    
    
    // where we're getting the row counts from
    auto const buf_index = (partition_index * src_bufs_per_partition) + d_metadata_col_to_buf_index[col_index];

    // start of the metadata buffer for this partition
    uint8_t* buf_start = out_buffer + out_buffer_offsets[partition_index];

    // start of row counts for this column
    size_type* row_count = reinterpret_cast<size_type*>(buf_start + (row_count_offsets[col_index] * sizeof(size_type)));
    if(d_col_data[col_index].type == cudf::type_id::STRING){
      row_count[0] = d_dst_buf_info[buf_index].root_num_rows;
      row_count[1] = d_dst_buf_info[buf_index].num_rows;      // # of chars
    } else {
      // all other columns write out just 1 row count. everything else can be reconstructed from there 
      // on the assemble side (eg offsets)
      row_count[0] = d_dst_buf_info[buf_index].num_rows;
    }

    /*
    bitmask_type mask = d_src_buf_info[buf_index].is_validity ? (1 << (col_index % 32)) : 0;
    auto const num_row_counts = row_count_offsets[metadata_size];
    bitmask_type* has_validity = reinterpret_cast<bitmask_type*>(buf_start + (num_row_counts * sizeof(size_type)));
    atomicOr(&has_validity[col_index / 32], mask);
    */
  });

  // compute the has-validity bits. this is being done as a full kernel because:
  // - we don't want to have to initialize the big rmm::device_buffer that contains all the partition data
  // - because of this, the value of all the has-validity bits will be random
  // - if we just used thrust, all we could do is call atomicOr which wouldn't handle the case where 
  //   the uninitialized memory is 1 but we want to set it to 0
#endif
}

};  // anonymous namespace

namespace detail {

/**
 * @brief A helper struct containing the state of contiguous_split, whether the caller
 * is using the single-pass contiguous_split or chunked_pack.
 *
 * It exposes an iterator-like pattern where contiguous_split_state::has_next()
 * returns true when there is work to be done, and false otherwise.
 *
 * contiguous_split_state::contiguous_split() performs a single-pass contiguous_split
 * and is valid iff contiguous_split_state is instantiated with 0 for the user_buffer_size.
 *
 * contiguous_split_state::contiguous_split_chunk(device_span) is only valid when
 * user_buffer_size > 0. It should be called as long as has_next() returns true. The
 * device_span passed to contiguous_split_chunk must be allocated in stream `stream` by
 * the user.
 *
 * None of the methods are thread safe.
 */
struct contiguous_split_state {
  contiguous_split_state(cudf::table_view const& input,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         std::optional<rmm::device_async_resource_ref> mr,
                         rmm::device_async_resource_ref temp_mr)
    : contiguous_split_state(input, {}, user_buffer_size, stream, mr, temp_mr)
  {
  }

  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         rmm::cuda_stream_view stream,
                         std::optional<rmm::device_async_resource_ref> mr,
                         rmm::device_async_resource_ref temp_mr)
    : contiguous_split_state(input, splits, 0, stream, mr, temp_mr)
  {
  }

  bool has_next() const { return !is_empty && chunk_iter_state->has_more_copies(); }

  std::size_t get_total_contiguous_size() const
  {
    return is_empty ? 0 : chunk_iter_state->total_size;
  }

  std::pair<shuffle_split_result, shuffle_split_metadata> contiguous_split()
  {
    CUDF_EXPECTS(user_buffer_size == 0, "Cannot contiguous split with a user buffer");
    if (is_empty || input.num_columns() == 0) { 
      return {shuffle_split_result{std::make_unique<rmm::device_buffer>(std::move(out_buffer)), std::move(out_buffer_offsets)},
              shuffle_split_metadata{std::move(metadata.global_metadata.col_info)}};
    }

    auto const num_batches_total =
      std::get<1>(chunk_iter_state->get_current_starting_index_and_buff_count());

    // perform the copy.
    copy_data(num_batches_total,
              0 /* starting at buffer for single-shot 0*/,
              src_and_dst_pointers->d_src_bufs,
              src_and_dst_pointers->d_dst_buf,
              chunk_iter_state->d_batched_dst_buf_info,
              nullptr,
              stream);

    // debug
    stream.synchronize();

    // these "orig" dst_buf_info pointers describe the prior-to-batching destination
    // buffers per partition
    auto d_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
    auto h_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;

    // postprocess valid_counts: apply the valid counts computed by copy_data for each
    // batch back to the original dst_buf_infos
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, out_to_in_index_function{chunk_iter_state->d_batch_offsets.begin(), (int)num_bufs});

    auto values = thrust::make_transform_iterator(
      chunk_iter_state->d_batched_dst_buf_info.begin(),
      cuda::proclaim_return_type<size_type>(
        [] __device__(dst_buf_info const& info) { return info.valid_count; }));

    thrust::reduce_by_key(rmm::exec_policy(stream, temp_mr),
                          keys,
                          keys + num_batches_total,
                          values,
                          thrust::make_discard_iterator(),
                          dst_valid_count_output_iterator{d_orig_dst_buf_info});

    CUDF_CUDA_TRY(hipMemcpyAsync(h_orig_dst_buf_info,
                                  d_orig_dst_buf_info,
                                  partition_buf_size_and_dst_buf_info->dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));

    stream.synchronize();

    // not necessary for the non-chunked case, but it makes it so further calls to has_next
    // return false, just in case
    chunk_iter_state->advance_iteration();
        
    // std::pair<shuffle_split_result, shuffle_split_metadata>
    return {shuffle_split_result{std::make_unique<rmm::device_buffer>(std::move(out_buffer)), std::move(out_buffer_offsets)},
            shuffle_split_metadata{std::move(metadata.global_metadata.col_info)}};
  }

  /*
  std::unique_ptr<std::vector<uint8_t>> build_packed_column_metadata()
  {
    CUDF_EXPECTS(num_partitions == 1, "build_packed_column_metadata supported only without splits");

    if (input.num_columns() == 0) { return std::unique_ptr<std::vector<uint8_t>>(); }

    if (is_empty) {
      // this is a bit ugly, but it was done to re-use make_empty_packed_table between the
      // regular contiguous_split and chunked_pack cases.
      auto empty_packed_tables = std::move(make_empty_packed_table().front());
      return std::move(empty_packed_tables.data.metadata);
    }

    auto& h_dst_buf_info  = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto cur_dst_buf_info = h_dst_buf_info;
    detail::metadata_builder mb{input.num_columns()};

    populate_metadata(input.begin(), input.end(), cur_dst_buf_info, mb);

    return std::make_unique<std::vector<uint8_t>>(std::move(mb.build()));
  }
  */

 private:
  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         std::optional<rmm::device_async_resource_ref> mr,
                         rmm::device_async_resource_ref temp_mr)
    : input(input),
      user_buffer_size(user_buffer_size),
      stream(stream),
      mr(mr),
      temp_mr(temp_mr),
      is_empty{check_inputs(input, splits)},
      num_partitions{splits.size() + 1},
      num_src_bufs{count_src_bufs(input.begin(), input.end())},
      num_bufs{num_src_bufs * num_partitions}
  {
    // compute metadata, even if the input is empty.
    metadata = compute_metadata(input);

    // if the table we are about to contig split is empty, no additional
    // work is necessary.
    if (is_empty) { return; }

    // debug
    stream.synchronize();

    // First pass over the source tables to generate a `dst_buf_info` per split and column buffer
    // (`num_bufs`). After this, contiguous_split uses `dst_buf_info` to further subdivide the work
    // into 1MB batches in `compute_batches`
    std::tie(partition_buf_size_and_dst_buf_info, partition_split_indices_and_src_buf_info) =
       compute_splits(input, splits, num_partitions, num_src_bufs, num_bufs, metadata, stream, temp_mr);

    // debug
    stream.synchronize();

    // generate output offsets from the partition buf sizes
    out_buffer_offsets = rmm::device_uvector<size_t>(num_partitions, stream, mr.value_or(cudf::get_current_device_resource()));
    auto size_iter = cudf::detail::make_counting_transform_iterator(0, partition_buf_size_func{{partition_buf_size_and_dst_buf_info->d_partition_sizes, num_partitions}});
    thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                           size_iter,
                           size_iter + num_partitions,
                           out_buffer_offsets.begin());

    // debug
    stream.synchronize();

    // one big output buffer
    out_buffer = rmm::device_buffer(partition_buf_size_and_dst_buf_info->h_dst_buf_total_size, stream, mr.value_or(cudf::get_current_device_resource()));

    /*
    void pack_per_partition_data(cz_metadata_internal const& metadata,
                             rmm::device_buffer& out_buffer,
                             rmm::device_uvector<size_t> const& out_buffer_offsets,
                             src_buf_info const* d_src_buf_info,
                             dst_buf_info const* d_dst_buf_info,
                             int src_bufs_per_partition,
                             size_type* d_metadata_col_to_buf_index,
                             rmm::cuda_stream_view stream)
                             */

    // pack the output metadata buffers
    pack_per_partition_data(metadata,
                            out_buffer,
                            out_buffer_offsets,
                            partition_split_indices_and_src_buf_info->d_src_buf_info,
                            partition_buf_size_and_dst_buf_info->d_dst_buf_info,
                            num_src_bufs,
                            partition_split_indices_and_src_buf_info->d_metadata_col_to_buf_index,
                            stream);

    // debug
    stream.synchronize();

    // Second pass: uses `dst_buf_info` to break down the work into 1MB batches.
    chunk_iter_state = compute_batches(num_bufs,
                                       partition_buf_size_and_dst_buf_info->d_dst_buf_info,
                                       partition_buf_size_and_dst_buf_info->h_dst_buf_total_size,
                                       num_partitions,
                                       user_buffer_size,
                                       stream,
                                       temp_mr);
    
    // debug
    stream.synchronize();
    
    CUDF_EXPECTS(user_buffer_size == 0, "Chunked mode not supported yet.");
    // allocate output partition buffers, in the non-chunked case
    /*
    if (user_buffer_size == 0) {
      out_buffers.reserve(num_partitions);
      auto h_buf_sizes = partition_buf_size_and_dst_buf_info->h_buf_sizes;
      std::transform(h_buf_sizes,
                     h_buf_sizes + num_partitions,
                     std::back_inserter(out_buffers),
                     [stream = stream,
                      mr = mr.value_or(rmm::mr::get_current_device_resource())](std::size_t bytes) {
                       return rmm::device_buffer{bytes, stream, mr};
                     });
    }
    */    

    src_and_dst_pointers = std::move(setup_src_and_dst_pointers(
      input, num_partitions, num_src_bufs, out_buffer, stream, temp_mr));
  }

  /*
  std::vector<packed_table> make_packed_tables()
  {
    if (input.num_columns() == 0) { return std::vector<packed_table>(); }
    if (is_empty) { return make_empty_packed_table(); }
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    std::vector<column_view> cols;
    cols.reserve(input.num_columns());

    auto& h_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto& h_dst_bufs     = src_and_dst_pointers->h_dst_bufs;

    auto cur_dst_buf_info = h_dst_buf_info;
    detail::metadata_builder mb(input.num_columns());

    for (std::size_t idx = 0; idx < num_partitions; idx++) {
      // traverse the buffers and build the columns.
      cur_dst_buf_info = build_output_columns(input.begin(),
                                              input.end(),
                                              cur_dst_buf_info,
                                              std::back_inserter(cols),
                                              h_dst_bufs[idx],
                                              mb);

      // pack the columns
      result.emplace_back(packed_table{
        cudf::table_view{cols},
        packed_columns{std::make_unique<std::vector<uint8_t>>(mb.build()),
                       std::make_unique<rmm::device_buffer>(std::move(out_buffers[idx]))}});

      cols.clear();
      mb.clear();
    }

    return result;
  }
  */

 /*
  std::vector<packed_table> make_empty_packed_table()
  {
    // sanitize the inputs (to handle corner cases like sliced tables)
    std::vector<cudf::column_view> empty_column_views;
    empty_column_views.reserve(input.num_columns());
    std::transform(input.begin(),
                   input.end(),
                   std::back_inserter(empty_column_views),
                   [](column_view const& col) { return cudf::empty_like(col)->view(); });

    table_view empty_inputs(empty_column_views);

    // build the empty results
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    auto const iter = thrust::make_counting_iterator(0);
    std::transform(iter,
                   iter + num_partitions,
                   std::back_inserter(result),
                   [&empty_inputs](int partition_index) {
                     return packed_table{empty_inputs,
                                         packed_columns{std::make_unique<std::vector<uint8_t>>(
                                                          pack_metadata(empty_inputs, nullptr, 0)),
                                                        std::make_unique<rmm::device_buffer>()}};
                   });

    return result;
  }
  */

  cudf::table_view const input;        ///< The input table_view to operate on
  std::size_t const user_buffer_size;  ///< The size of the user buffer for the chunked_pack case
  rmm::cuda_stream_view const stream;
  std::optional<rmm::device_async_resource_ref const> mr;  ///< The resource for any data returned

  // this resource defaults to `mr` for the contiguous_split case, but it can be useful for the
  // `chunked_pack` case to allocate scratch/temp memory in a pool
  rmm::device_async_resource_ref const temp_mr;  ///< The memory resource for scratch/temp space

  // whether the table was empty to begin with (0 rows or 0 columns) and should be metadata-only
  bool const is_empty;  ///< True if the source table has 0 rows or 0 columns

  // This can be 1 if `contiguous_split` is just packing and not splitting
  std::size_t const num_partitions;  ///< The number of partitions to produce

  size_type const num_src_bufs;  ///< Number of source buffers including children

  std::size_t const num_bufs;  ///< Number of source buffers including children * number of splits

  std::unique_ptr<packed_partition_buf_size_and_dst_buf_info>
    partition_buf_size_and_dst_buf_info;  ///< Per-partition buffer size and destination buffer info

  std::unique_ptr<packed_split_indices_and_src_buf_info>
    partition_split_indices_and_src_buf_info;  ///< Per-partition buffer size and destination buffer info

  std::unique_ptr<packed_src_and_dst_pointers>
    src_and_dst_pointers;  ///< Src. and dst. pointers for `copy_partition`

  //
  // State around the chunked pattern
  //

  // chunked_pack will have 1 or more "chunks" to iterate on, defined in chunk_iter_state
  // contiguous_split will have a single "chunk" in chunk_iter_state, so no iteration.
  std::unique_ptr<chunk_iteration_state>
    chunk_iter_state;  ///< State object for chunk iteration state

  // Two API usages are allowed:
  //  - `chunked_pack`: for this mode, the user will provide a buffer that must be at least 1MB.
  //    The behavior is "chunked" in that it will contiguously copy up until the user specified
  //    `user_buffer_size` limit, exposing a next() call for the user to invoke. Note that in this
  //    mode, no partitioning occurs, hence the name "pack".
  //
  //  - `contiguous_split` (default): when the user doesn't provide their own buffer,
  //    `contiguous_split` will allocate a buffer per partition and will place contiguous results in
  //    each buffer.
  //
  //std::vector<rmm::device_buffer>
//    out_buffers;  ///< Buffers allocated for a regular `contiguous_split`
  rmm::device_buffer              out_buffer{};
  rmm::device_uvector<size_t>     out_buffer_offsets{0, cudf::get_default_stream()};
  cz_metadata_internal metadata;  
};

};  // namespace detail

std::pair<shuffle_split_result, shuffle_split_metadata> shuffle_split(cudf::table_view const& input,
                                                                      std::vector<size_type> const& splits,
                                                                      rmm::cuda_stream_view stream,
                                                                      rmm::device_async_resource_ref mr)
{
  // for now, we don't allow strings, lists or columns with validity
  CUDF_EXPECTS(std::all_of(input.begin(), input.end(), [](cudf::column_view const& col){
    return col.type().id() != cudf::type_id::STRING && col.type().id() != cudf::type_id::LIST; }), "Unsupported column type (for now)");

  // `temp_mr` is the same as `mr` for contiguous_split as it allocates all
  // of its memory from the default memory resource in cuDF
  auto temp_mr = mr;
  auto state   = detail::contiguous_split_state(input, splits, stream, mr, temp_mr);
  return state.contiguous_split();
}

namespace detail {

#define OUTPUT_ITERATOR(__name, __T, __field_name)                                                  \
  template<typename __T>                                                                            \
  struct __name##generic_output_iter {                                                              \
    __T* c;                                                                                         \
    using value_type        = decltype(__T::__field_name);                                          \
    using difference_type   = size_t;                                                               \
    using pointer           = decltype(__T::__field_name)*;                                         \
    using reference         = decltype(__T::__field_name)&;                                         \
    using iterator_category = thrust::output_device_iterator_tag;                                   \
                                                                                                    \
    __name##generic_output_iter operator+ __host__ __device__(int i) { return {c + i}; }            \
                                                                                                    \
    __name##generic_output_iter& operator++ __host__ __device__()                                   \
    {                                                                                               \
      c++;                                                                                          \
      return *this;                                                                                 \
    }                                                                                               \
                                                                                                    \
    reference operator[] __device__(int i) { return dereference(c + i); }                           \
    reference operator* __device__() { return dereference(c); }                                     \
                                                                                                    \
  private:                                                                                          \
    reference __device__ dereference(__T* c) { return c->__field_name; }                            \
  };                                                                                                \
  using __name = __name##generic_output_iter<__T>

// per-flattened-column information
struct assemble_column_info {
  cudf::type_id         type;
  bool                  has_validity;
  size_type             num_rows, num_chars;
  size_type             null_count;
  size_type             num_children;
};
OUTPUT_ITERATOR(assemble_column_info_num_rows_output_iter, assemble_column_info, num_rows);
OUTPUT_ITERATOR(assemble_column_info_has_validity_output_iter, assemble_column_info, has_validity);

// a copy batch. 1 per block.
struct assemble_batch {
  __device__ assemble_batch(int8_t const* _src, int8_t* _dst, size_t _size, bool _validity, int _value_shift, int _bit_shift):
    src(_src), dst(_dst), size(_size), validity(_validity), value_shift(_value_shift), bit_shift(_bit_shift){}

  int8_t const* src;
  int8_t* dst;
  size_t              size;     // bytes
  bool                validity; // whether or not this is a validity buffer
  int value_shift;              // amount to shift values down by (for offset buffers)
  int bit_shift;                // # of bits to shift left by (for validity buffers)
  size_type valid_count = 0;    // (output) validity count for this block of work
};

struct assemble_column_functor {
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto const validity = buffer;
    auto const data = col->has_validity ? buffer + 1 : buffer;
    buffer = data + 1;

    out.push_back(std::make_unique<cudf::column>(cudf::data_type{col->type},
                  col->num_rows,
                  std::move(*data),
                  col->has_validity ? std::move(*validity) : rmm::device_buffer{},
                  col->null_count));
    
    return {col + 1, buffer};
  }
  
  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto const validity = buffer;
    buffer = col->has_validity ? buffer + 1 : buffer;

    // build children
    std::vector<std::unique_ptr<cudf::column>> children;
    children.reserve(col->num_children);
    auto next = col + 1;
    for(size_type i=0; i<col->num_children; i++){
      std::tie(next, buffer) = cudf::type_dispatcher(cudf::data_type{next->type},
                                                     detail::assemble_column_functor{stream, mr},
                                                     next,
                                                     buffer,
                                                     children);
    }    

    out.push_back(cudf::make_structs_column(col->num_rows,
                                            std::move(children),
                                            col->null_count,
                                            col->has_validity ? std::move(*validity) : rmm::device_buffer{},
                                            stream,
                                            mr));
    return {next, buffer};
  }
    
    /*
  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  size_t operator()(size_t cur, host_span<assemble_column_info const> assemble_data, host_span<rmm::device_buffer> buffers, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    auto col = assemble_data[cur];
    auto validity = cur;
    auto offsets = col.has_validity ? cur + 1 : cur;
    cur = offsets + 1;

    // build offsets
    auto offsets_col = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                      col.num_rows + 1,
                                                      std::move(buffers[offsets]),
                                                      rmm::device_buffer{},
                                                      0);

    // build the child
    std::vector<std::unique_ptr<cudf::column>> child_col;
    cur = cudf::type_dispatcher(cudf::data_type{col.type},
                                *this,
                                cur,
                                assemble_data,
                                buffers,
                                child_col);
    
    // build the final column
    out.push_back(cudf::make_lists_column(col.num_rows,
                                          std::move(offsets_col),
                                          std::move(child_col.back()),
                                          col.null_count,
                                          col.has_validity ? std::move(buffers[validity]) : rmm::device_buffer{},
                                          stream,
                                          mr));
    return cur;
  }  
  */

  // template <typename T, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::list_view> and !std::is_same_v<T, cudf::struct_view>)>
  template <typename T, typename ColumnIter, typename BufferIter, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>() and !std::is_same_v<T, cudf::struct_view>)>
  std::pair<ColumnIter, BufferIter> operator()(ColumnIter col, BufferIter buffer, std::vector<std::unique_ptr<cudf::column>>& out)
  {
    CUDF_FAIL("Unsupported type in shuffle_assemble");
  }
};

struct assemble_buffer_functor {
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  void operator()(assemble_column_info const& col, std::vector<rmm::device_buffer>& out)
  {
    // validity
    if(col.has_validity){
      out.push_back(alloc_validity(col.num_rows));
    }

    // data
    auto const data_size = cudf::util::round_up_safe(cudf::type_dispatcher(data_type{col.type}, size_of_helper{}) * col.num_rows, shuffle_split_partition_data_align);
    out.push_back(rmm::device_buffer(data_size, stream, mr));
  }

  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  void operator()(assemble_column_info const& col, std::vector<rmm::device_buffer>& out)
  { 
    // validity
    if(col.has_validity){
      out.push_back(alloc_validity(col.num_rows));
    }

    // offsets
    auto const offsets_size = cudf::util::round_up_safe(sizeof(size_type) * (col.num_rows + 1), shuffle_split_partition_data_align);
    out.push_back(rmm::device_buffer(offsets_size, stream, mr));
  } 

  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  void operator()(assemble_column_info const& col, std::vector<rmm::device_buffer>& out)
  { 
    // validity
    if(col.has_validity){
      out.push_back(alloc_validity(col.num_rows));
    }    
  }

  template <typename T, CUDF_ENABLE_IF(std::is_same_v<T, cudf::string_view>)>
  void operator()(assemble_column_info const& col, std::vector<rmm::device_buffer>& out)
  { 
    // validity
    if(col.has_validity){
      out.push_back(alloc_validity(col.num_rows));
    }

    // chars
    auto const chars_size = cudf::util::round_up_safe(sizeof(int8_t) * (col.num_chars + 1), shuffle_split_partition_data_align);
    out.push_back(rmm::device_buffer(chars_size, stream, mr));

    // offsets
    auto const offsets_size = cudf::util::round_up_safe(sizeof(size_type) * (col.num_rows + 1), shuffle_split_partition_data_align);
    out.push_back(rmm::device_buffer(offsets_size, stream, mr));
  }

  template <typename T, CUDF_ENABLE_IF(!std::is_same_v<T, cudf::struct_view> && 
                                       !std::is_same_v<T, cudf::list_view> && 
                                       !std::is_same_v<T, cudf::string_view> && 
                                       !cudf::is_fixed_width<T>())>
  void operator()(assemble_column_info const& col, std::vector<rmm::device_buffer>& out)
  { 
    CUDF_FAIL("Unsupported type in assemble_buffer_functor");
  }
 
private:
  rmm::device_buffer alloc_validity(size_type num_rows)
  {
    return rmm::device_buffer(bitmask_allocation_size_bytes(num_rows, shuffle_split_partition_data_align), stream, mr);
  }
};

// Computes required allocation size of a bitmask
__device__ std::size_t device_bitmask_allocation_size_bytes(size_type number_of_bits)
{
  auto const necessary_words = cudf::util::div_rounding_up_safe<size_type>(number_of_bits, 32);
  return necessary_words * sizeof(bitmask_type);
}

// Important: this returns the size of the buffer -without- padding. just the size of
// the raw bytes containing the actual data.
struct assemble_buffer_size_functor {
  template <typename T, typename OutputIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter out)
  {
    // validity
    if(col.has_validity){
      *out++ = device_bitmask_allocation_size_bytes(col.num_rows);
    }

    // data
    *out++ = cudf::type_dispatcher(data_type{col.type}, size_of_helper{}) * col.num_rows;
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out)
  { 
    // validity
    if(col.has_validity){
      *out++ = device_bitmask_allocation_size_bytes(col.num_rows);
    }

    // offsets
    *out++ = sizeof(size_type) * (col.num_rows + 1);
  } 

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out)
  { 
    // validity
    if(col.has_validity){
      *out++ = device_bitmask_allocation_size_bytes(col.num_rows);
    }
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::string_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out)
  { 
    // validity
    if(col.has_validity){
      *out++ = device_bitmask_allocation_size_bytes(col.num_rows);
    }

    // chars
    *out++ = sizeof(int8_t) * (col.num_chars + 1);

    // offsets
    *out++ = sizeof(size_type) * (col.num_rows + 1);
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(!std::is_same_v<T, cudf::struct_view> && 
                                       !std::is_same_v<T, cudf::list_view> && 
                                       !std::is_same_v<T, cudf::string_view> && 
                                       !cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter out)
  {
  }
};

struct assemble_metadata_offset_functor {
  template <typename T, typename OutputIter, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter out, size_t offset)
  {
    // validity
    if(col.has_validity){
      *out += offset;
      *out++;
    }

    // data
    *out += offset;
    out++;
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::list_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out, size_t offset)
  { 
    // validity
    if(col.has_validity){
      *out += offset;
      *out++;
    }

    // offsets
    (*out++) += offset;
    *out++;
  } 

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::struct_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out, size_t offset)
  { 
    // validity
    if(col.has_validity){
      *out += offset;
      *out++;
    }
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(std::is_same_v<T, cudf::string_view>)>
  __device__ void operator()(assemble_column_info const& col, OutputIter out, size_t offset)
  { 
    // validity
    if(col.has_validity){
      *out += offset;
      *out++;
    }

    // chars
    *out += offset;
    *out++;

    // offsets
    *out += offset;
    *out++;
  }

  template <typename T, typename OutputIter, CUDF_ENABLE_IF(!std::is_same_v<T, cudf::struct_view> && 
                                       !std::is_same_v<T, cudf::list_view> && 
                                       !std::is_same_v<T, cudf::string_view> && 
                                       !cudf::is_fixed_width<T>())>
  __device__ void operator()(assemble_column_info const& col, OutputIter out, size_t offset)
  {
  }
};

// returns:
// - a vector of assemble_column_info structs representing the destination column data.
//   the vector is of length global_metadata.col_info.size()  that is, the flattened list of columns in the table.
//
// - the same vector as above, but in host memory. 
//
// - a vector of assemble_column_info structs, representing the source column data.
//   the vector is of length global_metadata.col_info.size() * the # of partitions. 
//
std::tuple<rmm::device_uvector<assemble_column_info>,
           std::vector<assemble_column_info>,
           rmm::device_uvector<assemble_column_info>,
           size_t>
assemble_build_column_info(shuffle_split_metadata const& h_global_metadata,
                           cudf::device_span<int8_t const> partitions, 
                           cudf::device_span<size_t const> partition_offsets,
                           rmm::cuda_stream_view stream,
                           rmm::device_async_resource_ref mr)
{
  auto temp_mr = cudf::get_current_device_resource_ref();
  rmm::device_uvector<shuffle_split_col_data> global_metadata = cudf::detail::make_device_uvector_async(h_global_metadata.col_info, stream, temp_mr);

  // "columns" here means the number of flattened columns in the entire source table, not just the
  // number of columns at the top level
  auto const num_columns = global_metadata.size();
  size_type const num_partitions = partition_offsets.size();
  auto const num_column_instances = num_columns * num_partitions;

  // generate per-column data ------------------------------------------------------
  rmm::device_uvector<assemble_column_info> column_info(num_columns, stream, temp_mr);  

  // compute:
  //  - indices into the char count data for string columns
  //  - offset into the partition data where has-validity begins
  rmm::device_uvector<size_type> char_count_indices(num_columns + 1, stream, temp_mr);
  auto cc_index_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([global_metadata = global_metadata.begin(), num_columns] __device__ (size_type i) {
    return i >= num_columns ? 0 : (global_metadata[i].type == cudf::type_id::STRING ? 1 : 0);
  }));
  thrust::exclusive_scan(rmm::exec_policy_nosync(stream, temp_mr), cc_index_iter, cc_index_iter + num_columns + 1, char_count_indices.begin());
  size_type const per_partition_num_char_counts = char_count_indices.back_element(stream);
  // the +1 is for the per-partition overall row count at the very beginning
  auto const has_validity_offset = (per_partition_num_char_counts + 1) * sizeof(size_type);
  
  /*
  {
    auto h_char_count_indices = cudf::detail::make_std_vector_sync(char_count_indices, stream);
    printf("per_partition_num_char_counts : %d\n", per_partition_num_char_counts);
    printf("has_validity_offset : %lu\n", has_validity_offset);
    for(size_t idx=0; idx<h_char_count_indices.size(); idx++){
      printf("h_char_count_indices(%lu): %d\n", idx, h_char_count_indices[idx]);
    }
  } 
  */

  // compute has-validity
  // note that we are iterating vertically -> horizontally here, with each column's individual piece per partition first.
  auto column_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([num_partitions] __device__ (size_type i){
    return i / num_partitions;
  }));  
  auto has_validity_values = cudf::detail::make_counting_transform_iterator(0, 
    cuda::proclaim_return_type<bool>([num_partitions,
                                      has_validity_offset,
                                      partitions = partitions.data(),
                                      partition_offsets = partition_offsets.begin()]
                                      __device__ (int i) -> bool {
      auto const partition_index = i % num_partitions;
      bitmask_type const*const has_validity_buf = reinterpret_cast<bitmask_type const*>(partitions + partition_offsets[partition_index] + has_validity_offset);
      auto const col_index = i / num_partitions;
      // printf("HVV: %d, %d, %d, %d, %d\n", (int)partition_index, (int)partition_offsets[partition_index], (int)has_validity_offset, (int)col_index, (int)has_validity_buf[col_index / 32]);
      return has_validity_buf[col_index / 32] & (1 << (col_index % 32)) ? 1 : 0;
    })
  );
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        column_keys,
                        column_keys +  num_column_instances,
                        has_validity_values,
                        thrust::make_discard_iterator(),
                        assemble_column_info_has_validity_output_iter{column_info.begin()},
                        thrust::equal_to<size_type>{},
                        thrust::logical_or<bool>{});  
  /*
  {
    auto h_column_info = cudf::detail::make_std_vector_sync(column_info, stream);
    for(size_t idx=0; idx<h_column_info.size(); idx++){
      printf("h_column_info(%lu): has_validity = %d\n", idx, (int)(h_column_info[idx].has_validity ? 1 : 0));
    }
  } 
  */ 

  // print_span(cudf::device_span<size_t const>(partition_offsets));

  // compute overall row count
  auto row_count_values = cudf::detail::make_counting_transform_iterator(0,
    cuda::proclaim_return_type<cudf::size_type>([num_partitions,
                                                 partitions = partitions.data(),
                                                 partition_offsets = partition_offsets.begin()]
                                                 __device__ (int i){
                                                  return reinterpret_cast<size_type const*>(partitions + partition_offsets[i])[0];
                                                 }));
  size_t const row_count =  thrust::reduce(rmm::exec_policy_nosync(stream, temp_mr),
                                            row_count_values,
                                            row_count_values + num_partitions);
  
  // compute char counts for strings
  // note that we are iterating vertically -> horizontally here, with each column's individual piece per partition first.
  // TODO: use an output iterator and write directly to the outgoing assembly_info structs
  auto cc_keys = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<cudf::size_type>([num_partitions] __device__ (int i){
    return i / num_partitions;
  }));
  auto char_count_values = cudf::detail::make_counting_transform_iterator(0,
    cuda::proclaim_return_type<cudf::size_type>([num_partitions,
                                                 partitions = partitions.data(),
                                                 partition_offsets = partition_offsets.begin(),
                                                 global_metadata = global_metadata.begin()]
                                                 __device__ (int i){
      auto const partition_index = i % num_partitions;
      auto const col_index = i / num_partitions;

      // non-string columns don't have a char count
      auto const column_type = global_metadata[col_index].type;
      if(column_type != cudf::type_id::STRING){
        return 0;
      }

      // string columns
      size_type const*const char_counts = reinterpret_cast<size_type const*>(partitions + partition_offsets[partition_index] + 4);
      // printf("RCI %d : %d, partition_index = %d\n", (int)col_index, char_counts[col_index], (int)partition_index);
      return char_counts[col_index];
    })
  );
  rmm::device_uvector<size_type> char_counts(num_columns, stream, temp_mr);
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream, temp_mr),
                        cc_keys, 
                        cc_keys + num_column_instances,
                        char_count_values,
                        thrust::make_discard_iterator(),
                        char_counts.begin());
  // print_span(static_cast<cudf::device_span<size_type const>>(char_counts));
  
  // copy type and summed row counts
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(rmm::exec_policy_nosync(stream, temp_mr), iter, iter + num_columns, [row_count,
                                                                                        column_info = column_info.begin(),
                                                                                        global_metadata = global_metadata.begin(),
                                                                                        char_count_indices = char_count_indices.begin(),
                                                                                        char_counts = char_counts.begin()]
                                                                                        __device__ (size_type col_index){
    auto const& metadata = global_metadata[col_index];
    auto& cinfo = column_info[col_index];
    
    cinfo.type = metadata.type;
    cinfo.null_count = 0; // TODO
    cinfo.num_children = metadata.num_children;
    
    cinfo.num_rows = row_count;
    
    // string columns store the char count separately
    cinfo.num_chars = cinfo.type == cudf::type_id::STRING ? char_counts[char_count_indices[col_index]] : 0;
  });
    
  /*
  {
    auto h_column_info = cudf::detail::make_std_vector_sync(column_info, stream);
    for(size_t idx=0; idx<h_column_info.size(); idx++){
      printf("col_info[%lu]: type = %d has_validity = %d num_rows = %d num_chars = %d null_count = %d\n", idx,
        (int)h_column_info[idx].type, h_column_info[idx].has_validity ? 1 : 0, h_column_info[idx].num_rows, h_column_info[idx].num_chars, h_column_info[idx].null_count);
    }
  }
  */

  // generate per-column-instance data ------------------------------------------------------

  // has-validity, type, row count
  rmm::device_uvector<assemble_column_info> column_instance_info(num_column_instances, stream, temp_mr);
  thrust::for_each(rmm::exec_policy_nosync(stream, temp_mr), iter, iter + num_column_instances, [char_count_indices = char_count_indices.begin(),
                                                                                                 column_instance_info = column_instance_info.begin(),
                                                                                                 global_metadata = global_metadata.begin(),
                                                                                                 partitions = partitions.data(),
                                                                                                 partition_offsets = partition_offsets.begin(),
                                                                                                 num_columns,
                                                                                                 has_validity_offset]
                                                                                                 __device__ (size_type i){
    auto const partition_index = i / num_columns;
    auto const col_index = i % num_columns;
    auto const col_instance_index = (partition_index * num_columns) + col_index;

    auto const& metadata = global_metadata[col_index];
    auto& cinstance_info = column_instance_info[col_instance_index];

    uint8_t const*const pheader = reinterpret_cast<uint8_t const*>(partitions + partition_offsets[partition_index]);

    bitmask_type const*const has_validity_buf = reinterpret_cast<bitmask_type const*>(pheader + has_validity_offset);
    cinstance_info.has_validity = has_validity_buf[col_index / 32] & (1 << (col_index % 32)) ? 1 : 0;
    
    cinstance_info.type = metadata.type;
    cinstance_info.null_count = 0; // TODO
    cinstance_info.num_children = metadata.num_children;
    
    cinstance_info.num_rows = reinterpret_cast<size_type const*>(pheader)[0];
    
    // string columns store the char count separately
    if(metadata.type == cudf::type_id::STRING){
      size_type const*const char_counts = reinterpret_cast<size_type const*>(pheader + 4);
      cinstance_info.num_chars = char_counts[char_count_indices[col_index]];
    }
  });
    
  /*
  {
    auto h_column_instance_info = cudf::detail::make_std_vector_sync(column_instance_info, stream);
    for(size_t idx=0; idx<h_column_instance_info.size(); idx++){
      size_type const partition_index = idx / num_columns;
      size_type const col_index = idx % num_columns;
      size_type const col_instance_index = (partition_index * num_columns) + col_index;

      printf("col_info[%d, %d, %d]: type = %d has_validity = %d num_rows = %d num_chars = %d null_count = %d\n",
        partition_index, col_index, col_instance_index,
        (int)h_column_instance_info[idx].type, h_column_instance_info[idx].has_validity ? 1 : 0, h_column_instance_info[idx].num_rows, h_column_instance_info[idx].num_chars, h_column_instance_info[idx].null_count);
    }
  } 
  */

  // compute per-partition metadata size
  size_t const metadata_rc_size = ((per_partition_num_char_counts + 1) * sizeof(size_type));
  size_t const metadata_has_validity_size = (cudf::util::round_up_safe(num_columns, size_t{32}) / size_t{32}) * sizeof(bitmask_type);
  size_t const per_partition_metadata_size = cudf::util::round_up_safe(metadata_rc_size + metadata_has_validity_size, shuffle_split_partition_data_align);

  return {std::move(column_info), cudf::detail::make_std_vector_sync(column_info, stream), std::move(column_instance_info), per_partition_metadata_size};
}

template<typename SizeIterator, typename GroupFunction>
rmm::device_uvector<std::invoke_result_t<GroupFunction>> transform_expand(SizeIterator first,
                                                                          SizeIterator last,
                                                                          GroupFunction op,
                                                                          rmm::cuda_stream_view stream,
                                                                          rmm::device_async_resource_ref mr)
{ 
  auto temp_mr = cudf::get_current_device_resource_ref();

  auto value_count = std::distance(first, last);
  auto size_wrapper = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([value_count, first] __device__ (size_t i){
    return i >= value_count ? 0 : first[i];
  }));
  rmm::device_uvector<size_t> group_offsets(value_count + 1, stream, temp_mr);
  thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                         size_wrapper,
                         size_wrapper + group_offsets.size(),
                         group_offsets.begin());
  size_t total_size = group_offsets.back_element(stream); // note memcpy and device sync
  
  using OutputType = std::invoke_result_t<GroupFunction>;
  rmm::device_uvector<OutputType> result(total_size, stream, mr);
  auto iter = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + total_size,
                    result.begin(),
                    cuda::proclaim_return_type<OutputType>([op, group_offsets_begin = group_offsets.begin(), group_offsets_end = group_offsets.end()] __device__ (size_t i){
                      auto const group_index = thrust::lower_bound(thrust::seq, group_offsets_begin, group_offsets_end, i) - group_offsets_begin;
                      auto const intra_group_index = i - group_offsets_begin[group_index];
                      return op(group_index, intra_group_index);
                    }));

  return result;
}

// returns destination buffers
std::pair<std::vector<rmm::device_buffer>, rmm::device_uvector<assemble_batch>> assemble_build_buffers(rmm::device_uvector<assemble_column_info> const& column_info,
                                                                                                       rmm::device_uvector<assemble_column_info> const& column_instance_info,
                                                                                                       cudf::device_span<int8_t const> partitions,
                                                                                                       size_t num_partitions,
                                                                                                       size_t per_partition_metadata_size,
                                                                                                       rmm::cuda_stream_view stream,
                                                                                                       rmm::device_async_resource_ref mr)
{
  auto h_column_info = cudf::detail::make_std_vector_sync(column_info, stream);
  auto temp_mr = cudf::get_current_device_resource_ref();  
  
  // allocate output buffers ----------------------------------
  std::vector<rmm::device_buffer> assemble_buffers;
  assemble_buffers.reserve(h_column_info.size() * 3); // worst case, every column has 3 buffers.
  // mapping of column index to first-buffer index
  std::vector<size_type> h_column_to_buffer_map(h_column_info.size());  
  for(size_t idx=0; idx<h_column_info.size(); idx++){
    h_column_to_buffer_map[idx] = assemble_buffers.size();
    cudf::type_dispatcher(cudf::data_type{h_column_info[idx].type},
                          detail::assemble_buffer_functor{stream, mr},
                          h_column_info[idx],
                          assemble_buffers);
  }
  std::vector<int8_t*> h_dst_buffers(assemble_buffers.size());
  std::transform(assemble_buffers.begin(), assemble_buffers.end(), h_dst_buffers.begin(), [](rmm::device_buffer& buf){
    return reinterpret_cast<int8_t*>(buf.data());
  });
  auto dst_buffers = cudf::detail::make_device_uvector_async(h_dst_buffers, stream, temp_mr);
  auto column_to_buffer_map = cudf::detail::make_device_uvector_async(h_column_to_buffer_map, stream, cudf::get_current_device_resource_ref());
  // print_span(cudf::device_span<size_type const>{column_to_buffer_map});

  // generate copy batches ------------------------------------

  // compute:
  // - unpadded sizes of the source buffers
  // - offsets into the partition data where each source buffer starts
  // - offsets into the destination buffers where each source buffer starts writing
  // - starting row indices for each partition
  size_t const buffers_per_partition = assemble_buffers.size();
  size_t const num_src_buffers = buffers_per_partition * num_partitions;
  size_t const num_columns = column_info.size();
  rmm::device_uvector<size_t> src_sizes_unpadded(num_src_buffers, stream, mr);
  rmm::device_uvector<size_t> src_offsets(num_src_buffers, stream, mr);
  rmm::device_uvector<size_t> dst_offsets(num_src_buffers, stream, mr);
  rmm::device_uvector<size_type> partition_row_indices(num_partitions, stream, temp_mr);
  {
    // generate unpadded sizes of the source buffers
    auto const num_column_instances = column_instance_info.size();
    auto iter = thrust::make_counting_iterator(0);
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + num_column_instances,
                    [buffers_per_partition,
                     num_columns,
                     column_to_buffer_map = column_to_buffer_map.begin(),
                     column_instance_info = column_instance_info.begin(),
                     src_sizes_unpadded = src_sizes_unpadded.begin()] __device__ (size_type i){

      auto const partition_index = i / num_columns;
      auto const col_index = i % num_columns;
      auto const col_instance_index = (partition_index * num_columns) + col_index;

      auto const& cinfo_instance = column_instance_info[col_instance_index];
      auto const buf_index = column_to_buffer_map[col_index] + (partition_index * buffers_per_partition);
      cudf::type_dispatcher(cudf::data_type{cinfo_instance.type},
                            detail::assemble_buffer_size_functor{},
                            cinfo_instance,
                            &src_sizes_unpadded[buf_index]);
      // printf("SSU: %d %d (%d %d), %lu\n", (int)partition_index, (int)buf_index, (int)col_index, (int)column_to_buffer_map[col_index], src_sizes_unpadded[buf_index]);
    });
    // print_span(cudf::device_span<size_t const>{src_sizes_unpadded});
    
    // scan to source offsets. include padding for the buffers themselves
    auto padded_sizes = thrust::make_transform_iterator(src_sizes_unpadded.begin(), cuda::proclaim_return_type<size_t>([shuffle_split_partition_data_align = shuffle_split_partition_data_align] __device__ (size_t size_unpadded){
      return cudf::util::round_up_safe(size_unpadded, shuffle_split_partition_data_align);
    }));
    thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                           padded_sizes,
                           padded_sizes + src_sizes_unpadded.size(),
                           src_offsets.begin());
    // print_span(cudf::device_span<size_t const>{src_offsets});
    
    // add metadata header offset
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                    iter,
                    iter + num_column_instances,
                    [num_columns,
                     buffers_per_partition,
                     column_to_buffer_map = column_to_buffer_map.begin(),
                     column_instance_info = column_instance_info.begin(),
                     src_offsets = src_offsets.begin(),
                     per_partition_metadata_size] __device__ (size_type i){

      auto const partition_index = i / num_columns;
      auto const metadata_offset = (partition_index + 1) * per_partition_metadata_size;
      auto const col_index = i % num_columns;
      auto const col_instance_index = (partition_index * num_columns) + col_index;
      auto const& cinfo_instance = column_instance_info[col_instance_index];
      auto const buf_index = column_to_buffer_map[col_index] + (partition_index * buffers_per_partition);

      // printf("MHO: %d, partition_index = %d, metadata_offset = %d, col_index = %d, col_instance_index = %d, buf_index = %d\n", i, (int)partition_index, (int)metadata_offset, (int)col_index, (int)col_instance_index, (int)buf_index);
      cudf::type_dispatcher(cudf::data_type{cinfo_instance.type},
                            detail::assemble_metadata_offset_functor{},
                            cinfo_instance,
                            &src_offsets[buf_index],
                            metadata_offset);
    });
    // print_span(cudf::device_span<size_t const>{src_offsets});

    // generate destination buffer offsets
    // Note: vertical iteration
    auto dst_buf_key = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([num_partitions] __device__ (size_t i){
      return i / num_partitions;
    }));
    auto size_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_t>([src_sizes_unpadded = src_sizes_unpadded.begin(), num_partitions, buffers_per_partition] __device__ (size_t i){
      auto const dst_buf_index = i / num_partitions;
      auto const partition_index = i % num_partitions;
      auto const src_buf_index = (partition_index * buffers_per_partition) + dst_buf_index;
      return src_sizes_unpadded[src_buf_index];
    }));
    thrust::exclusive_scan_by_key(rmm::exec_policy(stream, temp_mr),
                                  dst_buf_key,
                                  dst_buf_key + num_src_buffers,
                                  size_iter,
                                  dst_offsets.begin());
    // print_span(cudf::device_span<size_t const>{dst_offsets});

    // for validity, we need to do a little more work. our destination positions are defined by bit position,
    // not byte position. so round down into the nearest starting bitmask word.    
    auto row_count_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<size_type>([column_instance_info = column_instance_info.begin(),
                                                                                                     columns_per_partition = column_info.size()] __device__ (size_type i){
      auto col = column_instance_info[i * columns_per_partition];
      return col.num_rows;
    }));
    thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                           row_count_iter,
                           row_count_iter + num_partitions,
                           partition_row_indices.begin());
    thrust::for_each(rmm::exec_policy(stream, temp_mr),
                     iter,
                     iter + num_column_instances,
                     [column_to_buffer_map = column_to_buffer_map.begin(),
                      column_info = column_info.begin(),
                      num_columns,
                      buffers_per_partition,
                      partition_row_indices = partition_row_indices.begin(),
                      dst_offsets = dst_offsets.begin()] __device__ (size_t i){
      auto const partition_index = i / num_columns;
      auto const col_index = i % num_columns;
      auto const& cinfo = column_info[col_index];
      if(cinfo.has_validity){
        auto const buf_index = column_to_buffer_map[col_index] + (partition_index * buffers_per_partition);
        dst_offsets[buf_index] = (partition_row_indices[partition_index] / 32) * sizeof(bitmask_type);
      }
    });
  }

  // generate batches
  auto batch_count_iter = cudf::detail::make_counting_transform_iterator(0, 
                                                                         cuda::proclaim_return_type<size_t>([src_sizes_unpadded = src_sizes_unpadded.begin()] __device__ (size_t i){
                                                                           return size_to_batch_count(src_sizes_unpadded[i]);
                                                                         }));
  auto copy_batches = transform_expand(batch_count_iter, 
                                       batch_count_iter + src_sizes_unpadded.size(),
                                       cuda::proclaim_return_type<assemble_batch>([dst_buffers = dst_buffers.begin(),
                                                                                   dst_offsets = dst_offsets.begin(),
                                                                                   partitions = partitions.data(),
                                                                                   buffers_per_partition,
                                                                                   num_partitions,
                                                                                   src_sizes_unpadded = src_sizes_unpadded.begin(),
                                                                                   src_offsets = src_offsets.begin(),
                                                                                   desired_batch_size = desired_batch_size,
                                                                                   partition_row_indices = partition_row_indices.begin()] __device__ (size_t src_buf_index, size_t batch_index){
                                         auto const batch_offset = batch_index * desired_batch_size;
                                         auto const partition_index = src_buf_index / buffers_per_partition;
                                         
                                         auto const src_offset = src_offsets[src_buf_index];
                                        
                                         auto const dst_buf_index = src_buf_index % buffers_per_partition;
                                         auto const dst_offset_index = (dst_buf_index * num_partitions) + partition_index;
                                         auto const dst_offset = dst_offsets[dst_offset_index];

                                         auto const bytes = std::min(src_sizes_unpadded[src_buf_index] - batch_offset, desired_batch_size);
                                         
                                         /*
                                         printf("ET: partition_index=%lu, src_buf_index=%lu, dst_buf_index=%lu, batch_index=%lu, src_offset=%lu, dst_offset=%lu bytes=%lu bit_shift = %d\n", 
                                           partition_index,
                                           src_buf_index,
                                           dst_buf_index,
                                           batch_index,
                                           src_offset + batch_offset,
                                           dst_offset + batch_offset,
                                           bytes,
                                           partition_row_indices[partition_index] % 32);
                                           */

                                         return assemble_batch {
                                          partitions + src_offset + batch_offset,
                                          dst_buffers[dst_buf_index] + dst_offset + batch_offset,
                                          bytes,
                                          0,  // TODO: handle offsets
                                          partition_row_indices[partition_index] % 32,  // bit shift for the validity copy step
                                          0};
                                         }),
                                       stream,
                                       mr);

  return {std::move(assemble_buffers), std::move(copy_batches)};
}

template<int block_size>
__global__ void copy_validity(cudf::device_span<assemble_batch> batches)
{
  int batch_index = blockIdx.x;
  auto& batch = batches[batch_index];
  
  __shared__ uint32_t next_word[block_size - 1];
  
  auto remaining_words = batch.size / sizeof(uint32_t);
  auto src = reinterpret_cast<bitmask_type const*>(batch.src);
  auto dst = reinterpret_cast<bitmask_type*>(batch.dst);   // dst starts at the first word corresponding to the row count of the first bit in src

  // eg, for a 3 bit shift:
  // upper_shift_mask = 0xd0000000
  // lower_shift_mask = 0x1fffffff
  auto const upper_shift = 32 - batch.bit_shift;
  uint32_t const upper_shift_mask = ((1 << batch.bit_shift) - 1) << upper_shift; 
  auto const lower_shift = batch.bit_shift;
  uint32_t const lower_shift_mask = ~upper_shift_mask;

  // read the first batch of words
  auto words_in_batch = min(static_cast<size_t>(block_size), remaining_words);
  uint32_t word = threadIdx.x < words_in_batch ? src[threadIdx.x] : 0;
  src += words_in_batch;
  remaining_words -= words_in_batch;

  // if we have a bit shift, use atomics to write the leading bits into the first word
  // which overlaps with another copy
  if(batch.bit_shift > 0){
    if(threadIdx.x == 0){
      atomicOr(dst, (word & upper_shift_mask) >> upper_shift);
    } else {
      next_word[threadIdx.x - 1] = word;
    }
    dst++;
    __syncthreads();
  }

  // copy the rest normally
  do {
    if(threadIdx.x < words_in_batch - 1){
      dst[threadIdx.x] = ((word & lower_shift_mask) << lower_shift) |
                         ((next_word[threadIdx.x] & upper_shift_mask) >> upper_shift);
    }
    if(remaining_words == 0){
      break;
    }

    // fetch next batch of words
    words_in_batch = min(static_cast<size_t>(block_size), remaining_words);
    word = threadIdx.x < words_in_batch ? src[threadIdx.x] : 0;
    next_word[threadIdx.x - 1] = word;
    src += words_in_batch;
    dst += words_in_batch;
    remaining_words -= words_in_batch;
    __syncthreads();
  } while(1);
  
  // trailing bits. always write this using atomicOr, even if we have no bit shift because we may not be writing a full
  // 32 bit word.
  if(threadIdx.x == words_in_batch - 1){
    atomicOr(dst + threadIdx.x, (word & lower_shift_mask) << lower_shift);
  }

  /*
  if(threadIdx.x == 0){
    batch.valid_count = valid_count;
  }
  */
}

void assemble_copy(rmm::device_uvector<assemble_batch> const& batches, rmm::cuda_stream_view stream)
{
  // main data copy. everything except validity and offsets
  {
    auto input_iter = thrust::make_transform_iterator(batches.begin(), cuda::proclaim_return_type<void*>([] __device__ (assemble_batch const& batch){
      return reinterpret_cast<void*>(const_cast<int8_t*>(batch.src));
    }));
    auto output_iter = thrust::make_transform_iterator(batches.begin(), cuda::proclaim_return_type<void*>([] __device__ (assemble_batch const& batch){
      return reinterpret_cast<void*>(batch.dst);
    }));
    auto size_iter = thrust::make_transform_iterator(batches.begin(), cuda::proclaim_return_type<size_t>([] __device__ (assemble_batch const& batch){
      // don't do any work for validity
      return batch.validity ? 0 : batch.size;
    }));

    size_t temp_storage_bytes;
    hipcub::DeviceMemcpy::Batched(nullptr, temp_storage_bytes, input_iter, output_iter, size_iter, batches.size(), stream);
    rmm::device_buffer temp_storage(temp_storage_bytes, stream, cudf::get_current_device_resource_ref());
    hipcub::DeviceMemcpy::Batched(temp_storage.data(), temp_storage_bytes, input_iter, output_iter, size_iter, batches.size(), stream);
  }

  // copy validity
  /*
  constexpr int block_size = 256;
  cudf::detail::grid_1d const grid{static_cast<cudf::thread_index_type>(batches.size()), block_size};
  copy_validity<block_size><<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(batches);
  */

  stream.synchronize();
}

// assemble all the columns and the final table from the intermediate buffers
std::unique_ptr<cudf::table> build_table(std::vector<assemble_column_info> const& assembly_data,
                                         std::vector<rmm::device_buffer>& assembly_buffers,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  std::vector<std::unique_ptr<cudf::column>> columns;
  auto column = assembly_data.begin();
  auto buffer = assembly_buffers.begin();  
  while(column != assembly_data.end()){    
    std::tie(column, buffer) = cudf::type_dispatcher(cudf::data_type{column->type},
                                                     detail::assemble_column_functor{stream, mr},
                                                     column,
                                                     buffer,
                                                     columns);
  }
  return std::make_unique<cudf::table>(std::move(columns));
}

} // namespace detail

std::unique_ptr<table> shuffle_assemble(shuffle_split_metadata const& global_metadata,
                                        cudf::device_span<int8_t const> partitions,
                                        cudf::device_span<size_t const> partition_offsets,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  // generate the info structs representing the flattened column hierarchy. the total number of assembled rows, null counts, etc
  auto [column_info, h_column_info, column_instance_info, per_partition_metadata_size] = detail::assemble_build_column_info(global_metadata, partitions, partition_offsets, stream, mr);

  // generate the (empty) output buffers based on the column info. note that is not a 1:1 mapping between column info
  // and buffers, since some columns will have validity and some will not.
  auto [dst_buffers, batches] = detail::assemble_build_buffers(column_info, column_instance_info, partitions, partition_offsets.size(), per_partition_metadata_size, stream, mr);  
  
  // copy the data. note that this does not sync.
  detail::assemble_copy(batches, stream);
  
  // return the final assembled table
  return build_table(h_column_info, dst_buffers, stream, mr);
}

};  // namespace spark_rapids_jni